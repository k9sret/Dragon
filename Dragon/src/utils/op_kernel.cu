
#include <hip/hip_runtime.h>
#ifdef WITH_CUDA

#include <cmath>

#include "core/context_cuda.h"
#include "core/tensor.h"
#include "utils/cuda_device.h"
#include "utils/op_kernel.h"
#include "utils/math_functions.h"
#include "utils/cast.h"

namespace dragon {

namespace kernel {

/******************** activation.dropout ********************/

template<typename T>
__global__ void _Dropout(
    const int               count,
    const uint32_t          thresh,
    const T                 scale,
    const T*                x,
    const uint32_t*         mask,
    T*                      y) {
    CUDA_1D_KERNEL_LOOP(idx, count) {
        y[idx] = x[idx] * (mask[idx] > thresh) * scale;
    }
}

template<> void Dropout<float, CUDAContext>(
    const int               count,
    float                   prob,
    float                   scale,
    const float*            x,
    uint32_t*               mask,
    float*                  y,
    CUDAContext*            ctx) {
    uint32_t thresh = static_cast<uint32_t>(UINT_MAX * prob);
    math::RandomUniform<uint32_t, CUDAContext>(
        count, float(0), float(UINT_MAX), mask, ctx);
    _Dropout<float>
        << < CUDA_BLOCKS(count), CUDA_THREADS,
             0, ctx->cuda_stream() >> >(
                 count, thresh, scale, x, mask, y);
}

template <typename T>
__global__ void _DropoutGrad(
    const int               count,
    const uint32_t          thresh,
    const T                 scale,
    const T*                dy, 
    const uint32_t*         mask,
    T*                      dx) {
    CUDA_1D_KERNEL_LOOP(idx, count) {
        dx[idx] = dy[idx] * (mask[idx] > thresh) * scale;
    }
}

template<> void DropoutGrad<float, CUDAContext>(
    const int               count,
    float                   prob,
    float                   scale,
    const float*            dy, 
    const uint32_t*         mask,
    float*                  dx,
    CUDAContext*            ctx) {
    uint32_t thresh = static_cast<uint32_t>(UINT_MAX * prob);
    _DropoutGrad<float>
        << < CUDA_BLOCKS(count), CUDA_THREADS,
             0, ctx->cuda_stream() >> >(
                 count, thresh, scale, dy, mask, dx);
}

/******************** activation.prelu ********************/

template <typename T>
__global__ void _PRelu(
    const int               count,
    const int               channels,
    const int               dim,
    const T*                x,
    const T*                w,
    T*                      y) {
    CUDA_1D_KERNEL_LOOP(idx, count) {
        y[idx] = (x[idx] > 0) * x[idx] +
            (x[idx] < 0) * x[idx] * w[0];
    }
}

template <typename T>
__global__ void _PReluNCHW(
    const int               count,
    const int               channels,
    const int               dim,
    const T*                x,
    const T*                w,
    T*                      y) {
    CUDA_1D_KERNEL_LOOP(idx, count) {
        const int c = (idx / dim) % channels;
        y[idx] = (x[idx] > 0) * x[idx] +
            (x[idx] < 0) * x[idx] * w[c];
    }
}

template <typename T>
__global__ void _PReluNHWC(
    const int               count,
    const int               channels,
    const int               dim,
    const T*                x,
    const T*                w,
    T*                      y) {
    CUDA_1D_KERNEL_LOOP(idx, count) {
        const int c = idx % channels;
        y[idx] = (x[idx] > 0) * x[idx] +
            (x[idx] < 0) * x[idx] * w[c];
    }
}

template<> void PRelu<float, CUDAContext>(const int count,
    const int               channels,
    const int               dim,
    const bool              channel_shared,
    const string&           data_format,
    const float*            x,
    const float*            w,
    float*                  y,
    CUDAContext*            ctx) {
    if (channel_shared) {
        _PRelu<float> 
            << < CUDA_BLOCKS(count), CUDA_THREADS,
                 0, ctx->cuda_stream() >> >(count,
                     channels, dim, x, w, y);
    } else {
        if (data_format == "NCHW") {
            _PReluNCHW<float>
                << < CUDA_BLOCKS(count), CUDA_THREADS,
                     0, ctx->cuda_stream() >> >(count,
                         channels, dim, x, w, y);
        } else if (data_format == "NHWC") {
            _PReluNHWC<float>
                << < CUDA_BLOCKS(count), CUDA_THREADS,
                     0, ctx->cuda_stream() >> >(count,
                         channels, dim, x, w, y);
        } else LOG(FATAL) << "Unknown data format: " << data_format;
    }
}

template <typename T>
__global__ void _PReluGrad(
    const int               count,
    const int               channels,
    const int               dim,
    const T*                dy,
    const T*                x,
    const T*                w,
    T*                      dx) {
    CUDA_1D_KERNEL_LOOP(idx, count) {
        dx[idx] = dy[idx] * (
            (x[idx] > 0) + (x[idx] <= 0) * w[0]
        );
    }
}

template <typename T>
__global__ void _PReluGradNCHW(
    const int               count,
    const int               channels,
    const int               dim,
    const T*                dy,
    const T*                x,
    const T*                w,
    T*                      dx) {
    CUDA_1D_KERNEL_LOOP(idx, count) {
        const int c = (idx / dim) % channels;
        dx[idx] = dy[idx] * (
            (x[idx] > 0) + (x[idx] <= 0) * w[c]
        );
    }
}

template <typename T>
__global__ void _PReluGradNHWC(
    const int               count,
    const int               channels,
    const int               dim,
    const T*                dy,
    const T*                x,
    const T*                w,
    T*                      dx) {
    CUDA_1D_KERNEL_LOOP(idx, count) {
        const int c = idx % channels;
        dx[idx] = dy[idx] * ((x[idx] > 0) + (x[idx] <= 0) * w[c]);
    }
}

template<> void PReluGrad<float, CUDAContext>(
    const int               count,
    const int               channels,
    const int               dim,
    const bool              channel_shared,
    const string&           data_format,
    const float*            dy,
    const float*            x,
    const float*            w,
    float*                  dx,
    CUDAContext*            ctx) {
    if (channel_shared) {
        _PReluGrad<float>
            << < CUDA_BLOCKS(count), CUDA_THREADS,
                 0, ctx->cuda_stream() >> >(count,
                     channels, dim, dy, x, w, dx);
    } else {
        if (data_format == "NCHW") {
            _PReluGradNCHW<float>
                << < CUDA_BLOCKS(count), CUDA_THREADS,
                     0, ctx->cuda_stream() >> >(count,
                         channels, dim, dy, x, w, dx);
        } else if (data_format == "NHWC") {
            _PReluGradNHWC<float>
                << < CUDA_BLOCKS(count), CUDA_THREADS,
                     0, ctx->cuda_stream() >> >(count,
                         channels, dim, dy, x, w, dx);
        } else LOG(FATAL) << "Unknown data format: " << data_format;
    }
}

template <typename T>
__global__ void _PReluWGradBcast(
    const int               count,
    const int               rows,
    const int               row_offset,
    const T*                dy,
    const T*                x,
    T*                      bcast_dw) {
    CUDA_1D_KERNEL_LOOP(idx, count) {
        bcast_dw[idx] = dy[idx] * x[idx] * (x[idx] <= 0);
        for (int n = 1; n < rows; n++) {
            const int cur_idx = idx + n * row_offset;
            bcast_dw[idx] +=
                dy[cur_idx] * x[cur_idx] * (x[cur_idx] <= 0);
        }
    }
}

template<> void PReluWGrad<float, CUDAContext>(
    const int               rows,
    const int               row_offset,
    const int               channels,
    const int               dim,
    const bool              channel_shared,
    const string&           data_format,
    const float*            dy,
    const float*            x,
    const float*            multiplier,
    float*                  bcast_dw,
    float*                  dw,
    CUDAContext*            ctx) {
    const int cdim = channels * dim;
    _PReluWGradBcast<float> 
        << < CUDA_BLOCKS(cdim), CUDA_THREADS,
             0, ctx->cuda_stream() >> >(
                 cdim, rows, row_offset, dy, x, bcast_dw);
    if (channel_shared) {
        float w_sum;
        math::Dot<float, CUDAContext>(channels * dim,
            bcast_dw, multiplier, &w_sum, ctx);
        math::AddScalar<float, CUDAContext>(1, w_sum, dw, ctx);
    } else {
        if (data_format == "NCHW") {
            math::Gemv<float, CUDAContext>(
                CblasNoTrans, channels, dim,
                1.0, bcast_dw, multiplier, 1.0, dw, ctx);
        } else if (data_format == "NHWC") {
            math::Gemv<float, CUDAContext>(
                CblasTrans, dim, channels,
                1.0, bcast_dw, multiplier, 1.0, dw, ctx);
        } else LOG(FATAL) << "Unknown data format: " << data_format;
    }
}

/******************** activation.elu ********************/

template <typename T>
__global__ void _Elu(
    const int               count,
    const T*                x,
    const float             alpha,
    T*                      y) {
    CUDA_1D_KERNEL_LOOP(idx, count) {
        y[idx] = x[idx] > 0 ? x[idx] :
            alpha * (exp(x[idx]) - 1);
    }
}

template<> void Elu<float, CUDAContext>(
    const int               count,
    const float             alpha,
    const float*            x,
    float*                  y,
    CUDAContext*            ctx) {
    _Elu<float>
        << < CUDA_BLOCKS(count), CUDA_THREADS,
             0, ctx->cuda_stream() >> >(count, x, alpha, y);
}

template <typename T>
__global__ void _EluGrad(
    const int               count,
    const float             alpha,
    const T*                dy,
    const T*                y,
    T*                      dx) {
    CUDA_1D_KERNEL_LOOP(idx, count) {
        dx[idx] = dy[idx] * (
            (y[idx] > 0) + (alpha + y[idx]) * (y[idx] <= 0)
        );
    }
}

template<> void EluGrad<float, CUDAContext>(
    const int               count,
    const float             alpha,
    const float*            dy,
    const float*            y,
    float*                  dx,
    CUDAContext*            ctx) {
    _EluGrad<float>
        << < CUDA_BLOCKS(count), CUDA_THREADS,
             0, ctx->cuda_stream() >> >(count, alpha, dy, y, dx);
}

/******************** activation.relu ********************/

template <typename T>
__global__ void _Relu(
    const int               count,
    const float             slope,
    const T*                x,
    T*                      y) {
    CUDA_1D_KERNEL_LOOP(idx, count) {
        y[idx] = x[idx] > 0 ? x[idx] : x[idx] * slope;
    }
}

template<> void Relu<float, CUDAContext>(
    const int               count,
    const float             slope,
    const float*            x,
    float*                  y,
    CUDAContext*            ctx) {
    _Relu<float>
        << < CUDA_BLOCKS(count), CUDA_THREADS,
             0, ctx->cuda_stream() >> >(count, slope, x, y);
}

template <typename T>
__global__ void _ReluGrad(
    const int               count,
    const float             slope,
    const T*                dy,
    const T*                y,
    T*                      dx) {
    CUDA_1D_KERNEL_LOOP(idx, count) {
        dx[idx] = dy[idx] * (
            (y[idx] > 0) + slope * (y[idx] <= 0)
        );
    }
}

template<> void ReluGrad<float, CUDAContext>(
    const int               count,
    const float             slope,
    const float*            dy,
    const float*            y,
    float*                  dx,
    CUDAContext*            ctx) {
    _ReluGrad<float> 
        << < CUDA_BLOCKS(count), CUDA_THREADS,
             0, ctx->cuda_stream() >> >(count, slope, dy, y, dx);
}

/******************** activation.selu ********************/

template <typename T>
__global__ void _SElu(
    const int               count,
    const T*                x,
    T*                      y) {
    CUDA_1D_KERNEL_LOOP(idx, count) {
        y[idx] = x[idx] > 0 ? 1.0507 * x[idx] :
            1.7581 * (exp(x[idx]) - 1);
    }
}

template<> void SElu<float, CUDAContext>(
    const int               count,
    const float*            x,
    float*                  y,
    CUDAContext*            ctx) {
    _SElu<float>
        << < CUDA_BLOCKS(count), CUDA_THREADS,
             0, ctx->cuda_stream() >> >(count, x, y);
}

template <typename T>
__global__ void _SEluGrad(
    const int               count,
    const T*                dy,
    const T*                y,
    T*                      dx) {
    CUDA_1D_KERNEL_LOOP(idx, count) {
        dx[idx] = y[idx] > 0 ? 1.0507 * dy[idx] :
            (1.7581 + y[idx]) * dy[idx];
    }
}

template<> void SEluGrad<float, CUDAContext>(
    const int               count,
    const float*            dy,
    const float*            y,
    float*                  dx,
    CUDAContext*            ctx) {
    _SEluGrad<float>
        << < CUDA_BLOCKS(count), CUDA_THREADS,
             0, ctx->cuda_stream() >> >(count, dy, y, dx);
}

/******************** activation.sigmoid ********************/

template <typename T>
__device__ T _SigmoidUnit(const T x) {
    return T(1) / (T(1) + exp(-x)); 
}

template <typename T>
__global__ void _Sigmoid(
    const int               n,
    const T*                x,
    T*                      y) {
    CUDA_1D_KERNEL_LOOP(idx, n) {
        y[idx] = _SigmoidUnit<T>(x[idx]);
    }
}

template<> void Sigmoid<float, CUDAContext>(
    const int               count,
    const float*            x,
    float*                  y,
    CUDAContext*            ctx) {
    _Sigmoid<float>
        << < CUDA_BLOCKS(count), CUDA_THREADS,
             0, ctx->cuda_stream() >> >(count, x, y);
}

template <typename T>
__global__ void _SigmoidGrad(
    const int               count,
    const T*                dy,
    const T*                y,
    T*                      dx) {
    CUDA_1D_KERNEL_LOOP(idx, count) {
        dx[idx] = dy[idx] * y[idx] * (1 - y[idx]);
    }
}

template<> void SigmoidGrad<float, CUDAContext>(
    const int               count,
    const float*            dy,
    const float*            y,
    float*                  dx,
    CUDAContext*            ctx) {
    _SigmoidGrad<float>
        << < CUDA_BLOCKS(count), CUDA_THREADS,
             0, ctx->cuda_stream() >> >(count, dy, y, dx);
}

/******************** activation.softmax ********************/

template <typename T>
__global__ void _SoftmaxMaxClass(
    const int               outer_dim,
    const int               classes,
    const int               inner_dim,
    const T*                x,
    T*                      scale) {
    CUDA_1D_KERNEL_LOOP(idx, outer_dim * inner_dim) {
        int o_idx = idx / inner_dim;
        int i_idx = idx % inner_dim;
        T max_val = -FLT_MAX;
        for (int c = 0; c < classes; c++)
            max_val = max(
                x[(o_idx * classes + c) * inner_dim + i_idx], max_val
            );
        scale[idx] = max_val;
    }
}

template <typename T>
__global__ void _SoftmaxSubtract(
    const int               count,
    const int               classes,
    const int               inner_dim,
    const T*                scale,
    T*                      y) {
    CUDA_1D_KERNEL_LOOP(idx, count) {
        int o_idx = idx / inner_dim / classes;
        int i_idx = idx % inner_dim;
        y[idx] -= scale[o_idx * inner_dim + i_idx];
    }
}

template <typename T>
__global__ void _SoftmaxExp(
    const int               count,
    T*                      y) {
    CUDA_1D_KERNEL_LOOP(idx, count) {
        y[idx] = exp(y[idx]);
    }
}

template <typename T>
__global__ void _SoftmaxSumClass(
    const int               outer_dim,
    const int               classes,
    const int               inner_dim,
    const T*                y,
    T* scale) {
    CUDA_1D_KERNEL_LOOP(idx, outer_dim * inner_dim) {
        int o_idx = idx / inner_dim;
        int i_idx = idx % inner_dim;
        T sum = 0;
        for (int c = 0; c < classes; c++)
            sum += y[(o_idx * classes + c) * inner_dim + i_idx];
        scale[idx] = sum;
    }
}

template <typename T>
 __global__ void _SoftmaxDiv(
     const int              count,
     const int              classes,
     const int              inner_dim,
     const T*               scale,
     T*                     y) {
    CUDA_1D_KERNEL_LOOP(idx, count) {
        int o_idx = idx / inner_dim / classes;
        int i_idx = idx % inner_dim;
        y[idx] /= scale[o_idx * inner_dim + i_idx];
    }
}

template<> void Softmax<float, CUDAContext>(
    const int               count,
    const int               classes,
    const int               outer_dim,
    const int               inner_dim,
    const float*            sum_multiplier,
    const float*            x,
    float*                  scale,
    float*                  y,
    CUDAContext*            ctx) {
    const int num_preds = inner_dim * outer_dim;
    _SoftmaxMaxClass<float>
        << < CUDA_BLOCKS(num_preds), CUDA_THREADS,
             0, ctx->cuda_stream() >> >(
                 outer_dim, classes, inner_dim, x, scale);
    _SoftmaxSubtract<float>
        << < CUDA_BLOCKS(count), CUDA_THREADS,
             0, ctx->cuda_stream() >> >(
                 count, classes, inner_dim, scale, y);
    _SoftmaxExp<float>
        << < CUDA_BLOCKS(count), CUDA_THREADS,
             0, ctx->cuda_stream() >> >(count, y);
    _SoftmaxSumClass<float>
        << < CUDA_BLOCKS(num_preds), CUDA_THREADS,
             0, ctx->cuda_stream() >> >(
                 outer_dim, classes, inner_dim, y, scale);
    _SoftmaxDiv<float>
        << < CUDA_BLOCKS(count), CUDA_THREADS,
             0, ctx->cuda_stream() >> >(
                count, classes, inner_dim, scale, y);
}

template <typename T>
__global__ void _SoftmaxDot(
    const int               outer_dim,
    const int               classes,
    const int               inner_dim,
    const T*                dy,
    const T*                y,
    T*                      scale) {
    CUDA_1D_KERNEL_LOOP(idx, outer_dim * inner_dim) {
        int o_idx = idx / inner_dim;
        int i_idx = idx % inner_dim;
        T dot = 0;
        for (int c = 0; c < classes; c++)
            dot += (
                y[(o_idx * classes + c) * inner_dim + i_idx] *
                    dy[(o_idx * classes + c) * inner_dim + i_idx]
            );
        scale[idx] = dot;
    }
}

template<> void SoftmaxGrad<float, CUDAContext>(
    const int               count,
    const int               classes,
    const int               outer_dim,
    const int               inner_dim,
    const float*            sum_multiplier,
    const float*            dy,
    const float*            y,
    float*                  scale,
    float*                  dx,
    CUDAContext*            ctx) {
    const int num_preds = inner_dim * outer_dim;
    _SoftmaxDot<float> 
        << < CUDA_BLOCKS(num_preds), CUDA_THREADS,
             0, ctx->cuda_stream() >> >(
                 outer_dim, classes, inner_dim, dy, y, scale);
    _SoftmaxSubtract<float> 
        << < CUDA_BLOCKS(count), CUDA_THREADS,
             0, ctx->cuda_stream() >> >(
                 count, classes,inner_dim, scale, dx);
    math::Mul<float, CUDAContext>(count, dx, y, dx, ctx);
}

/******************** activation.tanh ********************/

template <typename T>
__global__ void _Tanh(
    const int               count,
    const T*                x,
    T*                      y) {
    CUDA_1D_KERNEL_LOOP(i, count) {
        y[i] = tanh(x[i]);
    }
}

template<> void Tanh<float, CUDAContext>(
    const int               count,
    const float*            x,
    float*                  y,
    CUDAContext*            ctx) {
    _Tanh<float>
        << < CUDA_BLOCKS(count), CUDA_THREADS,
             0, ctx->cuda_stream() >> >(count, x, y);
}

template <typename T>
__global__ void _TanhGrad(
    const int               count,
    const T*                dy,
    const T*                y,
    T*                      dx) {
    CUDA_1D_KERNEL_LOOP(i, count) {
        dx[i] = dy[i] * (1 - y[i] * y[i]);
    }
}

template<> void TanhGrad<float, CUDAContext>(
    const int               count,
    const float*            dy,
    const float*            y,
    float*                  dx,
    CUDAContext*            ctx) {
    _TanhGrad<float>
        << < CUDA_BLOCKS(count), CUDA_THREADS,
             0, ctx->cuda_stream() >> >(count, dy, y, dx);
}

/******************** arithmetic.scale ********************/

template <typename T>
__global__ void _AffineWithOBias(
    const int               count,
    const int               scale_dim,
    const int               inner_dim,
    const T*                x,
    const T*                alpha,
    T*                      y) {
    CUDA_1D_KERNEL_LOOP(idx, count) {
        const int scale_idx = (idx / inner_dim) % scale_dim;
         y[idx] = alpha[scale_idx] * x[idx];
    }
}

template <typename T>
__global__ void _AffineWithBias(
    const int               count,
    const int               scale_dim,
    const int               inner_dim,
    const T*                x,
    const T*                alpha,
    const T*                beta,
    T*                      y) {
    CUDA_1D_KERNEL_LOOP(idx, count) {
        const int scale_idx = (idx / inner_dim) % scale_dim;
        y[idx] = alpha[scale_idx] * x[idx] + beta[scale_idx];
    }
}

template<> void Affine<float, CUDAContext>(
    const int               count,
    const int               outer_dim,
    const int               scale_dim,
    const int               inner_dim,
    const float*            x,
    const float*            alpha,
    const float*            beta,
    const float*            beta_multiplier,
    float*                  y,
    CUDAContext*            ctx) {
    if (beta != nullptr) {
        _AffineWithBias<float>
            << < CUDA_BLOCKS(count), CUDA_THREADS,
                 0, ctx->cuda_stream() >> >(
                     count, scale_dim, inner_dim,
                         x, alpha, beta, y);
    } else {
        _AffineWithOBias<float>
            << <CUDA_BLOCKS(count), CUDA_THREADS,
                0, ctx->cuda_stream() >> >(
                    count, scale_dim, inner_dim,
                        x, alpha, y);
    }
}

template <> void AffineGrad<float, CUDAContext>(
    const int               count,
    const int               outer_dim,
    const int               scale_dim,
    const int               inner_dim,
    const float*            dy,
    const float*            alpha,
    float*                  dx,
    CUDAContext*            ctx) {
    _AffineWithOBias<float>
        << < CUDA_BLOCKS(count), CUDA_THREADS,
             0, ctx->cuda_stream() >> >(
                 count, scale_dim, inner_dim,
                     dy, alpha, dx);
}

/******************** arithmetic.clip ********************/

template <typename T>
__global__ void _Clip(
    const int               count,
    const T                 low,
    const T                 high,
    const T*                x,
    T*                      mask,
    T*                      y) {
    CUDA_1D_KERNEL_LOOP(idx, count) {
        mask[idx] = 1.0;
        if (x[idx] > high || x[idx] < low) mask[idx] = 0.0;
        y[idx] = x[idx] > high ? high : x[idx];
        y[idx] = x[idx] < low ? low : x[idx];
    }
}

template <> void Clip<float, CUDAContext>(
    const int               count,
    const float             low,
    const float             high,
    const float*            x,
    float*                  mask,
    float*                  y,
    CUDAContext*            ctx) {
    _Clip<float>
        << < CUDA_BLOCKS(count), CUDA_THREADS,
             0, ctx->cuda_stream() >> >(count,
                 low, high, x, mask, y);
}

/******************** control_flow.compare ********************/

template <typename T>
__global__ void _Equal(
    const int               count,
    const T*                a,
    const T*                b,
    T*                      y) {
    CUDA_1D_KERNEL_LOOP(idx, count) {
        y[idx] = fabs(a[idx] - b[idx]) < FLT_EPSILON ? 1.0 : 0.0;
    }
}

template <> void Equal<float, CUDAContext>(
    const int               count,
    const float*            a,
    const float*            b,
    float*                  y,
    CUDAContext*            ctx) {
    _Equal<float>
        << < CUDA_BLOCKS(count), CUDA_THREADS,
             0, ctx->cuda_stream() >> >(count, a, b, y);
}

/******************** loss.l1_loss ********************/

template <typename T>
__global__ void _AbsGrad(
    const int               count,
    const T*                dy,
    T*                      dx) {
    CUDA_1D_KERNEL_LOOP(idx, count) {
       const T val = dy[idx];
       //  val > 0: 1 | val == 0: 0 | val < 0: -1
       dx[idx] = (val > T(0)) - (val < T(0));
    }
}

template<> void AbsGrad<float, CUDAContext>(
    const int               count,
    const float*            dy,
    float*                  dx,
    CUDAContext*            ctx) {
    _AbsGrad<float>
        << < CUDA_BLOCKS(count), CUDA_THREADS,
             0, ctx->cuda_stream() >> >(count, dy, dx);
}

/******************** loss.sigmoid_cross_entropy ********************/

template <typename T>
__global__ void _SigmoidCrossEntropy(
    const int               count,
    const T*                logits,
    const T*                targets,
    T*                      losses,
    T*                      flags) {
    CUDA_1D_KERNEL_LOOP(idx, count) {
        if (targets[idx] < 0) {
            losses[idx] = flags[idx] = 0;
        } else {
            losses[idx] = log(1 +
                exp(logits[idx] - 2 * logits[idx] * (logits[idx] >= 0))
            ) + logits[idx] * ((logits[idx] >= 0) - targets[idx]);
            flags[idx] = 1;
        }
    }
}

template <> void SigmoidCrossEntropy<float, CUDAContext>(
    const int               count,
    const float*            logits,
    const float*            targets,
    float*                  losses,
    float*                  flags,
    CUDAContext*            ctx) {
    _SigmoidCrossEntropy<float>
        << < CUDA_BLOCKS(count), CUDA_THREADS,
             0, ctx->cuda_stream() >> >(
                 count, logits, targets, losses, flags);
}

template <typename T>
__global__ void _SigmoidCrossEntropyGrad(
    const int               count,
    const T*                logits,
    const T*                targets,
    T*                      dlogits,
    T*                      flags) {
    CUDA_1D_KERNEL_LOOP(idx, count) {
        if (targets[idx] < 0) {
            dlogits[idx] = flags[idx] = 0;
        } else {
            dlogits[idx] = 1 / (1 + exp(-logits[idx])) - targets[idx];
            flags[idx] = 1;
        }
    }
}

template <> void SigmoidCrossEntropyGrad<float, CUDAContext>(
    const int               count,
    const float*            logits,
    const float*            targets,
    float*                  dlogits,
    float*                  flags,
    CUDAContext*            ctx) {
    _SigmoidCrossEntropyGrad<float>
        << < CUDA_BLOCKS(count), CUDA_THREADS,
             0, ctx->cuda_stream() >> >(
                 count, logits, targets, dlogits, flags);
}

/******************** loss.sigmoid_focal_loss ********************/

template <typename T>
__global__ void _SigmoidFocalLoss(
    const int               count,
    const int               axis_dim,
    const int               inner_dim,
    const float             pos_alpha,
    const float             neg_alpha,
    const float             gamma,
    const int               neg_id,
    const T*                logits,
    const T*                targets,
    T*                      losses,
    T*                      flags) {
    CUDA_1D_KERNEL_LOOP(idx, count) {
        const int iix = idx % inner_dim;
        const int aix = (idx / inner_dim) % axis_dim;
        const int oix = idx / inner_dim / axis_dim;
        const int t = targets[oix * inner_dim + iix];
        //  ``0`` is reserved for targets if neg id is zero
        //  use ``aix + 1`` to match the targets
        T c1 = (t == (aix + (neg_id ? 0 : 1)));
        T c2 = (t != -1) & (t != (aix + (neg_id ? 0 : 1)));
        T p = 1 / (1 + exp(-logits[idx]));  //  logit -> prob

        // (1 - p)^{gamma} * log(p)
        T pos_term = pow(1 - p, gamma) * log(max(p, FLT_MIN));

        // p^{gamma} * log(1 - p)
        T neg_term = pow(p, gamma) * (
            -logits[idx] * (logits[idx] >= 0) - log(
                1 + exp(logits[idx] - 2 * logits[idx] * (logits[idx] >= 0)))
       );

        losses[idx] = 0.0;
        losses[idx] += -c1 * pos_term * pos_alpha;
        losses[idx] += -c2 * neg_term * neg_alpha;
        flags[idx] = c1;
    }
}

template <> void SigmoidFocalLoss<float, CUDAContext>(
    const int               outer_dim,
    const int               axis_dim,
    const int               inner_dim,
    const float             pos_alpha,
    const float             neg_alpha,
    const float             gamma,
    const int               neg_id,
    const float*            logits,
    const float*            targets,
    float*                  losses,
    float*                  flags,
    CUDAContext*            ctx) {
    TIndex count = outer_dim * axis_dim * inner_dim;
    _SigmoidFocalLoss<float>
        << < CUDA_BLOCKS(count), CUDA_THREADS,
             0, ctx->cuda_stream() >> >(
                 count, axis_dim, inner_dim,
                     pos_alpha, neg_alpha, gamma, neg_id,
                         logits, targets, losses, flags);
}

template <typename T>
__global__ void _SigmoidFocalLossGradient(
    const int               count,
    const int               axis_dim,
    const int               inner_dim,
    const float             pos_alpha,
    const float             neg_alpha,
    const float             gamma,
    const int               neg_id,
    const T*                logits,
    const T*                targets,
    T*                      dlogits,
    T*                      flags) {
    CUDA_1D_KERNEL_LOOP(idx, count) {
        const int iix = idx % inner_dim;
        const int aix = (idx / inner_dim) % axis_dim;
        const int oix = idx / inner_dim / axis_dim;
        const int t = targets[oix * inner_dim + iix];
        //  ``0`` is reserved for targets if neg id is zero
        //  use ``aix + 1`` to match the targets
        T c1 = (t == (aix + (neg_id ? 0 : 1)));
        T c2 = (t != -1) & (t != (aix + (neg_id ? 0 : 1)));
        T p = 1 / (1 + exp(-logits[idx]));  //  logit -> prob

        // (1 - p)^{gamma} * (1 - p - gamma * p * log(p))
        T pos_term = pow((1 - p), gamma) * (
            1 - p - p * gamma * log(max(p, FLT_MIN))
        );

        // p^{gamma} * (gamma * (1 - p) * log(1-p) - p)
        T neg_term = pow(p, gamma) * (
            (-logits[idx] * (logits[idx] >= 0) - log(
                1 + exp(logits[idx] - 2 * logits[idx] * (logits[idx] >= 0)))
            ) * (1 - p) * gamma - p
        );

        dlogits[idx] = 0.0;
        dlogits[idx] += -c1 * pos_term * pos_alpha;
        dlogits[idx] += -c2 * neg_term * neg_alpha;
        flags[idx] = c1;
    }
}

template <> void SigmoidFocalLossGradient<float, CUDAContext>(
    const int               outer_dim,
    const int               axis_dim,
    const int               inner_dim,
    const float             pos_alpha,
    const float             neg_alpha,
    const float             gamma,
    const int               neg_id,
    const float*            logits,
    const float*            targets,
    float*                  dlogits,
    float*                  flags,
    CUDAContext*            ctx) {
    TIndex count = outer_dim * axis_dim * inner_dim;
    _SigmoidFocalLossGradient<float>
        << < CUDA_BLOCKS(count), CUDA_THREADS,
             0, ctx->cuda_stream() >> >(
                 count, axis_dim, inner_dim,
                     pos_alpha, neg_alpha, gamma, neg_id,
                         logits, targets, dlogits, flags);
}

/******************** loss.smooth_l1_loss ********************/

template <typename T>
__global__ void _SmoothL1(
    const int               count,
    const float             beta,
    const T*                x,
    T*                      y) {
    CUDA_1D_KERNEL_LOOP(idx, count) {
        const T val = x[idx];
        const T abs_val = abs(val);
        if (abs_val < beta) y[idx] = 0.5 * val * val / beta;
        else y[idx] = abs_val - 0.5 * beta;
    }
}

template<> void SmoothL1<float, CUDAContext>(
    const int               count,
    const float             beta,
    const float*            x,
    float*                  y,
    CUDAContext*            ctx) {
    _SmoothL1<float>
        << < CUDA_BLOCKS(count), CUDA_THREADS,
             0, ctx->cuda_stream() >> >(count, beta, x, y);
}

template <typename T>
__global__ void _SmoothL1Grad(
    const int               count,
    const float             beta,
    const T*                dy,
    T*                      dx) {
    CUDA_1D_KERNEL_LOOP(idx, count) {
        const T val = dy[idx];
        const T abs_val = abs(val);
        if (abs_val < beta) dx[idx] = val / beta;
        //  val > 0: 1 | val == 0: 0 | val < 0: -1
        else dx[idx] = (val > T(0)) - (val < T(0));
    }
}

template<> void SmoothL1Grad<float, CUDAContext>(
    const int               count,
    const float             beta,
    const float*            dy,
    float*                  dx,
    CUDAContext*            ctx) {
    _SmoothL1Grad<float>
        << < CUDA_BLOCKS(count), CUDA_THREADS,
             0, ctx->cuda_stream() >> >(count, beta, dy, dx);
}

/******************** loss.softmax_cross_entropy ********************/

template <typename T>
__global__ void _SoftmaxCrossEntropy(
    const int               count,
    const T*                prob,
    const T*                target,
    T*                      loss) {
    CUDA_1D_KERNEL_LOOP(idx, count) {
        loss[idx] = -target[idx] * log(max(prob[idx], FLT_MIN));
    }
}

template <> void SoftmaxCrossEntropy<float, CUDAContext>(
    const int               count,
    const float*            prob,
    const float*            target,
    float*                  loss,
    CUDAContext*            ctx) {
    _SoftmaxCrossEntropy<float>
        << < CUDA_BLOCKS(count), CUDA_THREADS,
             0, ctx->cuda_stream() >> >(count, prob, target, loss);
}

/******************** loss.softmax_focal_loss ********************/

template <typename T>
__global__ void _SoftmaxFocalLoss(
    const int               count,
    const int               axis_dim,
    const int               inner_dim,
    const float             pos_alpha,
    const float             neg_alpha,
    const float             gamma,
    const int               neg_id,
    const T*                prob,
    const T*                labels,
    const int*              ignores,
    const int               num_ignores,
    T*                      losses,
    T*                      flags) {
    CUDA_1D_KERNEL_LOOP(idx, count) {
        const int oix = idx / inner_dim;
        const int iix = idx % inner_dim;
        const int label = labels[oix * inner_dim + iix];
        int k;
        for (k = 0; k < num_ignores; k++) {
            if (label == ignores[k]) {
                losses[idx] = flags[idx] = 0;
                break;
            }
        }
        if (k == num_ignores) {
            const int t = (oix * axis_dim + label) * inner_dim + iix;
            T scale = pow(1.f - prob[t], gamma);
            scale = label > neg_id ?
                pos_alpha * scale : neg_alpha * scale;
            losses[idx] = -scale * log(max(prob[t], FLT_MIN));
            flags[idx] = label > neg_id ? 1 : 0;
        }
    }
}

template <> void SoftmaxFocalLoss<float, CUDAContext>(
    const int               outer_dim,
    const int               axis_dim,
    const int               inner_dim,
    const float             pos_alpha,
    const float             neg_alpha,
    const float             gamma,
    const int               neg_id,
    const float*            prob,
    const float*            labels,
    const int*              ignores,
    const int               num_ignores,
    float*                  losses,
    float*                  flags,
    CUDAContext*            ctx) {
    const int num_preds = outer_dim * inner_dim;
    _SoftmaxFocalLoss<float>
        << < CUDA_BLOCKS(num_preds), CUDA_THREADS,
             0, ctx->cuda_stream() >> >(
                 num_preds, axis_dim, inner_dim,
                     pos_alpha, neg_alpha, gamma, neg_id,
                         prob, labels, ignores, num_ignores,
                             losses, flags);
}

template <typename T>
__global__ void _SoftmaxFocalLossGrad(
    const int               count,
    const int               axis_dim,
    const int               inner_dim,
    const float             pos_alpha,
    const float             neg_alpha,
    const float             gamma,
    const int               neg_id,
    const T*                prob,
    const T*                labels,
    const int*              ignores,
    const int               num_ignores,
    T*                      dx,
    T*                      flags) {
    CUDA_1D_KERNEL_LOOP(idx, count) {
        const int oix = idx / inner_dim;
        const int iix = idx % inner_dim;
        const int label = labels[oix * inner_dim + iix];
        int k;
        for (k = 0; k < num_ignores; k++)
            if (label == ignores[k]) break;
        if (k != num_ignores) {
            for (int c = 0; c < axis_dim; c++)
                dx[(oix * axis_dim + c) * inner_dim + iix] = 0;
            flags[idx] = 0;
        } else {
            const int t = (oix * axis_dim + label) * inner_dim + iix;
            T onemp = 1. - prob[t];
            //  unstable if gamma is 0
            T grad = -gamma * pow(onemp, gamma - 1)
                            * log(max(prob[t], FLT_MIN))
                            * prob[t] + pow(onemp, gamma);
            grad = label > neg_id ?
                pos_alpha * grad : neg_alpha * grad;
            for (int c = 0; c < axis_dim; c++) {
                const int i = (oix * axis_dim + c) * inner_dim + iix;
                if (c == label) {
                    dx[i] = grad * (prob[t] - 1);
                } else {
                    dx[i] = grad * prob[i];
                }
            }
            flags[idx] = label > neg_id ? 1 : 0;
        }
    }
}

template<> void SoftmaxFocalLossGrad<float, CUDAContext>(
    const int               outer_dim,
    const int               axis_dim,
    const int               inner_dim,
    const float             pos_alpha,
    const float             neg_alpha,
    const float             gamma,
    const int               neg_id,
    const float*            prob,
    const float*            labels,
    const int*              ignores,
    const int               num_ignores,
    float*                  dx,
    float*                  flags,
    CUDAContext*            ctx) {
    const int num_preds = outer_dim * inner_dim;
    _SoftmaxFocalLossGrad<float>
        << < CUDA_BLOCKS(num_preds), CUDA_THREADS,
             0, ctx->cuda_stream() >> >(
                 num_preds, axis_dim, inner_dim,
                     pos_alpha, neg_alpha, gamma, neg_id,
                         prob, labels, ignores, num_ignores,
                             dx, flags);
}

/******************** loss.sparse_softmax_cross_entropy ********************/

template <typename Tx, typename Ty>
__global__ void _SparseSoftmaxCrossEntropy(
    const int               count,
    const int               axis_dim,
    const int               inner_dim,
    const Tx*               prob,
    const Ty*               labels,
    const int*              ignores,
    const int               num_ignores,
    Tx*                     losses,
    Tx*                     flags) {
    CUDA_1D_KERNEL_LOOP(idx, count) {
        const int oix = idx / inner_dim;
        const int iix = idx % inner_dim;
        const int label = labels[oix * inner_dim + iix];
        int k;
        for (k = 0; k < num_ignores; k++) {
            if (label == ignores[k]) {
                losses[idx] = flags[idx] = 0;
                break;
            }
        }
        if (k == num_ignores) {
            losses[idx] = -log(
                max(prob[(oix * axis_dim + label)
                    * inner_dim + iix], FLT_MIN)
            );
            flags[idx] = 1;
        }
    }
}

template <> void SparseSoftmaxCrossEntropy<float, float, CUDAContext>(
    const int               outer_dim,
    const int               axis_dim,
    const int               inner_dim,
    const float*            prob,
    const float*            labels,
    const int*              ignores,
    const int               num_ignores,
    float*                  losses,
    float*                  flags,
    CUDAContext*            ctx) {
    const int num_preds = outer_dim * inner_dim;
    _SparseSoftmaxCrossEntropy<float, float>
        << < CUDA_BLOCKS(num_preds), CUDA_THREADS,
             0, ctx->cuda_stream() >> >(
                 num_preds, axis_dim, inner_dim,
                     prob, labels, ignores, num_ignores,
                         losses, flags);
}

template <> void SparseSoftmaxCrossEntropy<float, int64_t, CUDAContext>(
    const int               outer_dim,
    const int               axis_dim,
    const int               inner_dim,
    const float*            prob,
    const int64_t*          labels,
    const int*              ignores,
    const int               num_ignores,
    float*                  losses,
    float*                  flags,
    CUDAContext*            ctx) {
    const int num_preds = outer_dim * inner_dim;
    _SparseSoftmaxCrossEntropy<float, int64_t>
        << < CUDA_BLOCKS(num_preds), CUDA_THREADS,
             0, ctx->cuda_stream() >> >(
                 num_preds, axis_dim, inner_dim,
                     prob, labels, ignores, num_ignores,
                         losses, flags);
}

template <typename Tx, typename Ty>
__global__ void _SparseSoftmaxCrossEntropyGrad(
    const int               count,
    const int               axis_dim,
    const int               inner_dim,
    const Tx*               prob,
    const Ty*               labels,
    const int*              ignores,
    const int               num_ignores,
    Tx*                     dx,
    Tx*                     flags) {
    CUDA_1D_KERNEL_LOOP(idx, count) {
        const int oix = idx / inner_dim;
        const int iix = idx % inner_dim;
        const int label = labels[oix * inner_dim + iix];
        int k;
        for (k = 0; k < num_ignores; k++)
                if (label == ignores[k]) break;
        if (k != num_ignores) {
            for (int c = 0; c < axis_dim; c++)
                dx[(oix * axis_dim + c) * inner_dim + iix] = 0;
            flags[idx] = 0;
        } else {
            dx[(oix * axis_dim + label) * inner_dim + iix] -= 1;
            flags[idx] = 1;
        }
    }
}

template<> void SparseSoftmaxCrossEntropyGrad<float, float, CUDAContext>(
    const int               outer_dim,
    const int               axis_dim,
    const int               inner_dim,
    const float*            prob,
    const float*            labels,
    const int*              ignores,
    const int               num_ignores,
    float*                  dx,
    float*                  flags,
    CUDAContext*            ctx) {
    const int num_preds = outer_dim * inner_dim;
    _SparseSoftmaxCrossEntropyGrad<float, float>
        << < CUDA_BLOCKS(num_preds), CUDA_THREADS,
             0, ctx->cuda_stream() >> >(
                 num_preds, axis_dim, inner_dim,
                     prob, labels, ignores, num_ignores,
                         dx, flags);
}

template<> void SparseSoftmaxCrossEntropyGrad<float, int64_t, CUDAContext>(
    const int               outer_dim,
    const int               axis_dim,
    const int               inner_dim,
    const float*            prob,
    const int64_t*          labels,
    const int*              ignores,
    const int               num_ignores,
    float*                  dx,
    float*                  flags,
    CUDAContext*            ctx) {
    const int num_preds = outer_dim * inner_dim;
    _SparseSoftmaxCrossEntropyGrad<float, int64_t>
        << < CUDA_BLOCKS(num_preds), CUDA_THREADS,
             0, ctx->cuda_stream() >> >(
                 num_preds, axis_dim, inner_dim,
                     prob, labels, ignores, num_ignores,
                         dx, flags);
}

/******************** misc.astype ********************/

template <typename Ta, typename Tb>
__global__ void _TypeA2B(
    const int               count,
    const Ta*               a,
    Tb*                     b) {
    CUDA_1D_KERNEL_LOOP(idx, count) {
        b[idx] = a[idx];
    }
}

#define DEFINE_TYPE_A2B(type_a, type_b) \
    template <> void TypeA2B<type_a, type_b, CUDAContext>( \
        const int           count, \
        const               type_a* a, \
        type_b*             b, \
        CUDAContext*        ctx) { \
        _TypeA2B<type_a, type_b> \
            << < CUDA_BLOCKS(count), CUDA_THREADS, \
                 0, ctx->cuda_stream() >> >(count, a, b); \
    }

#define DEFINE_TYPE_A2ALL(type_a) \
    DEFINE_TYPE_A2B(type_a, float); \
    DEFINE_TYPE_A2B(type_a, double); \
    DEFINE_TYPE_A2B(type_a, int); \
    DEFINE_TYPE_A2B(type_a, int64_t); \
    DEFINE_TYPE_A2B(type_a, uint8_t);

DEFINE_TYPE_A2ALL(float);
DEFINE_TYPE_A2ALL(double);
DEFINE_TYPE_A2ALL(int);
DEFINE_TYPE_A2ALL(int64_t);
DEFINE_TYPE_A2ALL(uint8_t);

/******************** misc.image_data ********************/

template <typename Tx, typename Ty>
__global__ void _ImageData_NCHW(
    const int               count,
    const int               N,
    const int               C,
    const int               H,
    const int               W,
    const float*            mean_values,
    const float*            std_values,
    const Tx*               x,
    Ty*                     y) {
    CUDA_1D_KERNEL_LOOP(idx, count) {
        const int w = idx % W;
        const int h = (idx / W) % H;
        const int c = (idx / W / H) % C;
        const int n = idx / W / H / C;
        Ty raw_value = x[((n * H + h) * W + w) * C + c];
        if (mean_values != nullptr) raw_value -= mean_values[c];
        if (std_values != nullptr) raw_value /= std_values[c];
        y[idx] = raw_value;
    }
}

template <typename Tx, typename Ty>
__global__ void _ImageData_NHWC(
    const int               count,
    const int               N,
    const int               C,
    const int               H,
    const int               W,
    const float*            mean_values,
    const float*            std_values,
    const Tx*               x,
    Ty*                     y) {
    CUDA_1D_KERNEL_LOOP(idx, count) {
        const int c = idx % C;
        Ty raw_value = x[idx];
        if (mean_values != nullptr) raw_value -= mean_values[c];
        if (std_values != nullptr) raw_value /= std_values[c];
        y[idx] = raw_value;
    }
}

template <> void ImageData<float, float, CUDAContext>(
    const int               count,
    const int               N,
    const int               C,
    const int               H,
    const int               W,
    const float*            mean_values,
    const float*            std_values,
    const string&           data_format,
    const float*            x,
    float*                  y,
    CUDAContext*            ctx) {
    if (data_format == "NCHW") {
        _ImageData_NCHW<float, float>
            << < CUDA_BLOCKS(count), CUDA_THREADS,
                 0, ctx->cuda_stream() >> >(count,
                     N, C, H, W, mean_values, std_values, x, y);
    } else if (data_format == "NHWC") {
        _ImageData_NHWC<float, float>
            << < CUDA_BLOCKS(count), CUDA_THREADS,
                 0, ctx->cuda_stream() >> >(count,
                     N, C, H, W, mean_values, std_values, x, y);
    } else LOG(FATAL) << "Unknown data format: " << data_format;
}

template <> void ImageData<uint8_t, float, CUDAContext>(
    const int               count,
    const int               N,
    const int               C,
    const int               H,
    const int               W,
    const float*            mean_values,
    const float*            std_values,
    const string&           data_format,
    const uint8_t*          x,
    float*                  y,
    CUDAContext*            ctx) {
    if (data_format == "NCHW") {
        _ImageData_NCHW<uint8_t, float>
            << < CUDA_BLOCKS(count), CUDA_THREADS,
                 0, ctx->cuda_stream() >> >(count,
                     N, C, H, W, mean_values, std_values, x, y);
    } else if (data_format == "NHWC") {
        _ImageData_NHWC<uint8_t, float>
            << < CUDA_BLOCKS(count), CUDA_THREADS,
                 0, ctx->cuda_stream() >> >(count,
                     N, C, H, W, mean_values, std_values, x, y);
    } else LOG(FATAL) << "Unknown data format: " << data_format;
}

/******************** ndarray.arange ********************/

template <typename T>
__global__ void _Arange(
    const int               count,
    const int               start,
    const int               step,
    T*                      y) {
    CUDA_1D_KERNEL_LOOP(idx, count) {
        y[idx] = start + idx * step;
    }
}

template<> void Arange<float, CUDAContext>(
    const int               count,
    const int               start,
    const int               step,
    float*                  y,
    CUDAContext*            ctx) {
    _Arange<float>
        << < CUDA_BLOCKS(count), CUDA_THREADS,
             0, ctx->cuda_stream() >> >(count, start, step, y);
}

template<> void Arange<int, CUDAContext>(
    const int               count,
    const int               start,
    const int               step,
    int*                    y,
    CUDAContext*            ctx) {
    _Arange<int>
        << < CUDA_BLOCKS(count), CUDA_THREADS,
             0, ctx->cuda_stream() >> >(count, start, step, y);
}

/******************** ndarray.argreduce ********************/

template <typename T>
__global__ void _Argmax(
    const int               count,
    const int               axis_dim,
    const int               inner_dim,
    const T                 neg_bound,
    const T*                x,
    int64_t*                indices) {
    CUDA_1D_KERNEL_LOOP(idx, count) {
        const int oix = idx / inner_dim;
        const int iix = idx % inner_dim;
        int max_idx = -1; T max_val = neg_bound;
        for (int j = 0; j < axis_dim; ++j) {
            const T val = x[(oix * axis_dim + j)
                                 * inner_dim + iix];
            if (val > max_val) {
                max_val = val;
                max_idx = j;
            }
        }
        indices[idx] = max_idx;
    }
}

template <typename T>
__global__ void _Argmax_v2(
    const int               count,
    const int               axis_dim,
    const int               inner_dim,
    const T                 neg_bound,
    const T*                x,
    int64_t*                indices,
    T*                      values) {
    CUDA_1D_KERNEL_LOOP(idx, count) {
        const int oix = idx / inner_dim;
        const int iix = idx % inner_dim;
        int max_idx = -1; T max_val = neg_bound;
        for (int j = 0; j < axis_dim; ++j) {
            const T val = x[(oix * axis_dim + j)
                                 * inner_dim + iix];
            if (val > max_val) {
                max_val = val;
                max_idx = j;
            }
        }
        indices[idx] = max_idx;
        values[idx] = max_val;
    }
}

template<> void Argmax<float, CUDAContext>(
    const int               count,
    const int               axis_dim,
    const int               inner_dim,
    const int               top_k,
    const float*            x,
    int64_t*                indices,
    float*                  values,
    CUDAContext*            ctx) {
    CHECK_EQ(top_k, 1) << "top_k > 1 is not supported with CUDA";
    if (values == nullptr) {
        _Argmax<float>
            << < CUDA_BLOCKS(count), CUDA_THREADS,
                 0, ctx->cuda_stream() >> >(count,
                     axis_dim, inner_dim, -FLT_MAX,
                         x, indices);
    } else {
        _Argmax_v2<float>
            << < CUDA_BLOCKS(count), CUDA_THREADS,
                 0, ctx->cuda_stream() >> >(count,
                     axis_dim, inner_dim, -FLT_MAX,
                         x, indices, values);
    }
}

template <typename T>
__global__ void _Argmin(
    const int               count,
    const int               axis_dim,
    const int               inner_dim,
    const T                 pos_bound,
    const T*                x,
    int64_t*                indices) {
    CUDA_1D_KERNEL_LOOP(idx, count) {
        const int oix = idx / inner_dim;
        const int iix = idx % inner_dim;
        int min_idx = -1; T min_val = pos_bound;
        for (int j = 0; j < axis_dim; ++j) {
            const T val = x[(oix * axis_dim + j)
                                 * inner_dim + iix];
            if (val < min_val) {
                min_val = val;
                min_idx = j;
            }
        }
        indices[idx] = min_idx;
    }
}

template <typename T>
__global__ void _Argmin_v2(
    const int               count,
    const int               axis_dim,
    const int               inner_dim,
    const T                 pos_bound,
    const T*                x,
    int64_t*                indices,
    T*                      values) {
    CUDA_1D_KERNEL_LOOP(idx, count) {
        const int oix = idx / inner_dim;
        const int iix = idx % inner_dim;
        int min_idx = -1; T min_val = pos_bound;
        for (int j = 0; j < axis_dim; ++j) {
            const T val = x[(oix * axis_dim + j)
                                 * inner_dim + iix];
            if (val < min_val) {
                min_val = val;
                min_idx = j;
            }
        }
        indices[idx] = min_idx;
        values[idx] = min_val;
    }
}

template<> void Argmin<float, CUDAContext>(
    const int               count,
    const int               axis_dim,
    const int               inner_dim,
    const int               top_k,
    const float*            x,
    int64_t*                indices,
    float*                  values,
    CUDAContext*            ctx) {
    CHECK_EQ(top_k, 1) << "top_k > 1 is not supported with CUDA";
    if (values == nullptr) {
        _Argmin<float>
            << < CUDA_BLOCKS(count), CUDA_THREADS,
                 0, ctx->cuda_stream() >> >(count,
                     axis_dim, inner_dim, FLT_MAX,
                         x, indices);
    } else {
        _Argmin_v2<float>
            << < CUDA_BLOCKS(count), CUDA_THREADS,
                 0, ctx->cuda_stream() >> >(count,
                     axis_dim, inner_dim, FLT_MAX,
                         x, indices, values);
    }
}

/******************** ndarray.gather ********************/

template <typename T>
__global__ void _CanonicalAxis(
    const int               count,
    const int               dim,
    T*                      y) {
    CUDA_1D_KERNEL_LOOP(idx, count) {
        if (y[idx] < 0) y[idx] += dim;
    }
}

template <> void CanonicalAxis<int, CUDAContext>(
    const int               count,
    const int               dim,
    int*                    y,
    CUDAContext*            ctx) {
    _CanonicalAxis<int>
        << < CUDA_BLOCKS(count), CUDA_THREADS,
             0, ctx->cuda_stream() >> >(count, dim, y);
}

template <typename T>
__global__ void _Gather(
    const int               count,
    const int               outer_dim,
    const int               inner_dim,
    const int               x_slice_dim,
    const int               y_slice_dim,
    const int*              indices,
    const T*                x,
    T*                      y) {
    CUDA_1D_KERNEL_LOOP(idx, count) {
        const int outer_idx = idx / inner_dim / y_slice_dim;
        const int slice_idx = idx % inner_dim;
        const int y_idx_offset = (idx / inner_dim) % y_slice_dim;
        const int x_idx_offset = indices[y_idx_offset];
        const int x_idx = (outer_idx * x_slice_dim + x_idx_offset)
                                     * inner_dim + slice_idx;
        y[idx] = x[x_idx];
    }
}

template <> void Gather<float, CUDAContext>(
    const int               count,
    const int               outer_dim,
    const int               inner_dim,
    const int               x_slice_dim,
    const int               y_slice_dim,
    const int*              indices,
    const float*            x,
    float*                  y,
    CUDAContext*            ctx) {
    _Gather<float>
        << <CUDA_BLOCKS(count), CUDA_THREADS,
            0, ctx->cuda_stream() >> >(count,
                outer_dim, inner_dim,
                    x_slice_dim, y_slice_dim,
                        indices, x, y);
}

template <> void Gather<int, CUDAContext>(
    const int               count,
    const int               outer_dim,
    const int               inner_dim,
    const int               x_slice_dim,
    const int               y_slice_dim,
    const int*              indices,
    const int*              x,
    int*                    y,
    CUDAContext*            ctx) {
    _Gather<int>
        << <CUDA_BLOCKS(count), CUDA_THREADS,
            0, ctx->cuda_stream() >> >(count,
                outer_dim, inner_dim,
                    x_slice_dim, y_slice_dim,
                        indices, x, y);
}

template <typename T>
__global__ void _GatherGrad(
    const int               count,
    const int               outer_dim,
    const int               inner_dim,
    const int               x_slice_dim,
    const int               y_slice_dim,
    const int*              indices,
    const T*                dy,
    T*                      dx) {
    CUDA_1D_KERNEL_LOOP(idx, count) {
        const int outer_idx = idx / inner_dim / y_slice_dim;
        const int slice_idx = idx % inner_dim;
        const int y_idx_offset = (idx / inner_dim) % y_slice_dim;
        const int x_idx_offset = indices[y_idx_offset];
        const int x_idx = (outer_idx * x_slice_dim + x_idx_offset)
                                     * inner_dim + slice_idx;
        atomicAdd(dx + x_idx, dy[idx]);
    }
}

template <> void GatherGrad<float, CUDAContext>(
    const int               count,
    const int               outer_dim,
    const int               inner_dim,
    const int               x_slice_dim,
    const int               y_slice_dim,
    const int*              indices,
    const float*            dy,
    float*                  dx,
    CUDAContext*            ctx) {
    _GatherGrad<float>
        << <CUDA_BLOCKS(count), CUDA_THREADS,
            0, ctx->cuda_stream() >> >(count,
                outer_dim, inner_dim,
                    x_slice_dim, y_slice_dim,
                        indices, dy, dx);
}

template <> void GatherGrad<int, CUDAContext>(
    const int               count,
    const int               outer_dim,
    const int               inner_dim,
    const int               x_slice_dim,
    const int               y_slice_dim,
    const int*              indices,
    const int*              dy,
    int*                    dx,
    CUDAContext*            ctx) {
    _GatherGrad<int>
        << <CUDA_BLOCKS(count), CUDA_THREADS,
            0, ctx->cuda_stream() >> >(count,
                outer_dim, inner_dim,
                    x_slice_dim, y_slice_dim,
                        indices, dy, dx);
}

/******************** ndarray.concat ********************/

template <typename T>
__global__ void _Concat(
    const int               count,
    const int               outer_dim,
    const int               inner_dim,
    const int               x_concat_dim,
    const int               y_concat_dim,
    const int               concat_offset,
    const T*                x,
    T*                      y) {
    CUDA_1D_KERNEL_LOOP(idx, count) {
        const int tmp = x_concat_dim * inner_dim;
        const int outer_idx = idx / tmp;
        const int concat_idx = idx % tmp;
        const int y_idx = (outer_idx * y_concat_dim + concat_offset)
                                     * inner_dim + concat_idx;
        y[y_idx] = x[idx];
    }
}

template <> void Concat<float, CUDAContext>(
    const int               count,
    const int               outer_dim,
    const int               inner_dim,
    const int               x_concat_dim,
    const int               y_concat_dim,
    const int               concat_offset,
    const float*            x,
    float*                  y,
    CUDAContext*            ctx) {
    _Concat<float>
        << < CUDA_BLOCKS(count), CUDA_THREADS,
             0, ctx->cuda_stream() >> >(count,
                 outer_dim, inner_dim,
                     x_concat_dim, y_concat_dim,
                         concat_offset, x, y);
}

template <typename T>
__global__ void _ConcatGrad(
    const int               count,
    const int               outer_dim,
    const int               inner_dim,
    const int               x_concat_dim,
    const int               y_concat_dim,
    const int               concat_offset,
    const T*                dy,
    T*                      dx) {
    CUDA_1D_KERNEL_LOOP(idx, count) {
        const int tmp = x_concat_dim * inner_dim;
        const int outer_idx = idx / tmp;
        const int concat_idx = idx % tmp;
        const int y_idx = (outer_idx * y_concat_dim + concat_offset)
                                     * inner_dim + concat_idx;
        dx[idx] = dy[y_idx];
    }
}

template <> void ConcatGrad<float, CUDAContext>(
    const int               count,
    const int               outer_dim,
    const int               inner_dim,
    const int               x_concat_dim,
    const int               y_concat_dim,
    const int               concat_offset,
    const float*            dy,
    float*                  dx,
    CUDAContext*            ctx) {
    _ConcatGrad<float>
        << < CUDA_BLOCKS(count), CUDA_THREADS,
             0, ctx->cuda_stream() >> >(count,
                 outer_dim, inner_dim,
                     x_concat_dim, y_concat_dim,
                         concat_offset, dy, dx);
}

/******************** ndarray.crop ********************/

template<typename T>
__global__ void _Crop1D(
    const int               count,
    const int               dim,
    const int               ex_dim,
    const int               inner_dim,
    const int               start,
    const T*                x,
    T*                      y) {
    CUDA_1D_KERNEL_LOOP(idx, count) {
        const int i = idx % inner_dim;
        const int ex_d = (idx / inner_dim) % ex_dim;
        const int o = idx / inner_dim / ex_dim;
        y[idx] = x[(o * dim + ex_d + start) * inner_dim + i];
    }
}

template<> void Crop1D<int, CUDAContext>(
    const int               count,
    const int               dim,
    const int               ex_dim,
    const int               inner_dim,
    const int               start,
    const int*              x,
    int*                    y,
    CUDAContext*            ctx) {
    _Crop1D<int>
        << < CUDA_BLOCKS(count), CUDA_THREADS,
             0, ctx->cuda_stream() >> >(count,
                 dim, ex_dim, inner_dim, start, x, y);
}

template<> void Crop1D<float, CUDAContext>(
    const int               count,
    const int               dim,
    const int               ex_dim,
    const int               inner_dim,
    const int               start,
    const float*            x,
    float*                  y,
    CUDAContext*            ctx) {
    _Crop1D<float>
        << < CUDA_BLOCKS(count), CUDA_THREADS,
             0, ctx->cuda_stream() >> >(count,
                 dim, ex_dim, inner_dim, start, x, y);
}

template<typename T>
__global__ void _Crop1DGrad(
    const int               count,
    const int               dim,
    const int               ex_dim,
    const int               inner_dim,
    const int               start,
    const int               end,
    const T*                dy,
    T*                      dx) {
    CUDA_1D_KERNEL_LOOP(idx, count) {
        const int i = idx % inner_dim;
        const int d = (idx / inner_dim) % dim;
        const int o = idx / inner_dim / dim;
        dx[idx] = (d < start || d >= end) ? 0 :
            dy[(o * ex_dim + d - start) * inner_dim + i];
    }
}

template<> void Crop1DGrad<int, CUDAContext>(
    const int               count,
    const int               dim,
    const int               ex_dim,
    const int               inner_dim,
    const int               start,
    const int               end,
    const int*              dy,
    int*                    dx,
    CUDAContext*            ctx) {
    _Crop1DGrad<int>
        << < CUDA_BLOCKS(count), CUDA_THREADS,
             0, ctx->cuda_stream() >> >(count,
                 dim, ex_dim, inner_dim, start, end, dy, dx);
}

template<> void Crop1DGrad<float, CUDAContext>(
    const int               count,
    const int               dim,
    const int               ex_dim,
    const int               inner_dim,
    const int               start,
    const int               end,
    const float*            dy,
    float*                  dx,
    CUDAContext*            ctx) {
    _Crop1DGrad<float>
        << < CUDA_BLOCKS(count), CUDA_THREADS,
             0, ctx->cuda_stream() >> >(count,
                 dim, ex_dim, inner_dim, start, end, dy, dx);
}

/******************** ndarray.pad ********************/

template <typename T>
__global__ void _ConstPad1D(
    const int               count,
    const int               dim,
    const int               ex_dim,
    const int               inner_dim,
    const int               pad_l,
    const T                 value,
    const T*                x,
    T*                      y) {
    CUDA_1D_KERNEL_LOOP(idx, count) {
        const int i = idx % inner_dim;
        const int ex_d = (idx / inner_dim) % ex_dim;
        const int o = idx / inner_dim / ex_dim;
        const int d = ex_d - pad_l;
        y[idx] = (d < 0 || d >= dim) ? value :
            x[(o * dim + d) * inner_dim + i];
    }
}

template <> void ConstPad1D<float, CUDAContext>(
    const int               count,
    const int               dim,
    const int               ex_dim,
    const int               inner_dim,
    const int               pad_l,
    const float             value,
    const float*            x,
    float*                  y,
    CUDAContext*            ctx) {
    _ConstPad1D<float>
        << < CUDA_BLOCKS(count), CUDA_THREADS,
             0, ctx->cuda_stream() >> >(count,
                 dim, ex_dim, inner_dim, pad_l, value, x, y);
}

template <typename T>
__global__ void _ReflectPad1D(
    const int               count,
    const int               dim,
    const int               ex_dim,
    const int               inner_dim,
    const int               pad_l,
    const T*                x,
    T*                      y) {
    CUDA_1D_KERNEL_LOOP(idx, count) {
        const int i = idx % inner_dim;
        const int ex_d = (idx / inner_dim) % ex_dim;
        const int o = idx / inner_dim / ex_dim;
        int d = ex_d - pad_l;
        d = max(d, -d);
        d = min(d, 2 * dim - d - 2);
        y[idx] = x[(o * dim + d) * inner_dim + i];
    }
}

template <> void ReflectPad1D<float, CUDAContext>(
    const int               count,
    const int               dim,
    const int               ex_dim,
    const int               inner_dim,
    const int               pad_l,
    const float*            x,
    float*                  y,
    CUDAContext*            ctx) {
    _ReflectPad1D<float>
        << < CUDA_BLOCKS(count), CUDA_THREADS,
             0, ctx->cuda_stream() >> >(count,
                 dim, ex_dim, inner_dim, pad_l, x, y);
}

template <typename T>
__global__ void _EdgePad1D(
    const int               count,
    const int               dim,
    const int               ex_dim,
    const int               inner_dim,
    const int               pad_l,
    const T*                x,
    T*                      y) {
    CUDA_1D_KERNEL_LOOP(idx, count) {
        const int i = idx % inner_dim;
        const int ex_d = (idx / inner_dim) % ex_dim;
        const int o = idx / inner_dim / ex_dim;
        const int d = min(dim - 1, max(ex_d - pad_l, 0));
        y[idx] = x[(o * dim + d) * inner_dim + i];
    }
}

template <> void EdgePad1D<float, CUDAContext>(
    const int               count,
    const int               dim,
    const int               ex_dim,
    const int               inner_dim,
    const int               pad_l,
    const float*            x,
    float*                  y,
    CUDAContext*            ctx) {
    _EdgePad1D<float>
        << < CUDA_BLOCKS(count), CUDA_THREADS,
             0, ctx->cuda_stream() >> >(count,
                 dim, ex_dim, inner_dim, pad_l, x, y);
}

template <typename T>
__global__ void _ConstPad1DGrad(
    const int               count,
    const int               dim,
    const int               ex_dim,
    const int               inner_dim,
    const int               pad_l,
    const T*                dy,
    T*                      dx) {
    CUDA_1D_KERNEL_LOOP(idx, count) {
        const int i = idx % inner_dim;
        const int ex_d = (idx / inner_dim) % dim + pad_l;
        const int o = idx / inner_dim / dim;
        dx[idx] = dy[(o * ex_dim + ex_d) * inner_dim + i];
    }
}

template <> void ConstPad1DGrad<float, CUDAContext>(
    const int               count,
    const int               dim,
    const int               ex_dim,
    const int               inner_dim,
    const int               pad_l,
    const float*            dy,
    float*                  dx,
    CUDAContext*            ctx) {
    _ConstPad1DGrad<float>
        << < CUDA_BLOCKS(count), CUDA_THREADS,
             0, ctx->cuda_stream() >> >(count,
                 dim, ex_dim, inner_dim, pad_l, dy, dx);
}

template <typename T>
__global__ void _ReflectPad1DGrad(
    const int               count,
    const int               dim,
    const int               ex_dim,
    const int               inner_dim,
    const int               pad_l,
    const T*                dy,
    T*                      dx) {
    CUDA_1D_KERNEL_LOOP(idx, count) {
        const int i = idx % inner_dim;
        const int ex_d = (idx / inner_dim) % ex_dim;
        const int o = idx / inner_dim / ex_dim;
        int d = ex_d - pad_l;
        d = max(d, -d);
        d = min(d, 2 * dim - d - 2);
        atomicAdd(&dx[(o * dim + d) * inner_dim + i], dy[idx]);
    }
}

template <> void ReflectPad1DGrad<float, CUDAContext>(
    const int               count,
    const int               dim,
    const int               ex_dim,
    const int               inner_dim,
    const int               pad_l,
    const float*            dy,
    float*                  dx,
    CUDAContext*            ctx) {
    _ReflectPad1DGrad<float>
        << < CUDA_BLOCKS(count), CUDA_THREADS,
             0, ctx->cuda_stream() >> >(count,
                 dim, ex_dim, inner_dim, pad_l, dy, dx);
}

template <typename T>
__global__ void _EdgePad1DGrad(
    const int               count,
    const int               dim,
    const int               ex_dim,
    const int               inner_dim,
    const int               pad_l,
    const T*                dy,
    T*                      dx) {
    CUDA_1D_KERNEL_LOOP(idx, count) {
        const int i = idx % inner_dim;
        const int ex_d = (idx / inner_dim) % ex_dim;
        const int o = idx / inner_dim / ex_dim;
        const int d = min(dim - 1, max(ex_d - pad_l, 0));
        atomicAdd(&dx[(o * dim + d) * inner_dim + i], dy[idx]);
    }
}

template <> void EdgePad1DGrad<float, CUDAContext>(
    const int               count,
    const int               dim,
    const int               ex_dim,
    const int               inner_dim,
    const int               pad_l,
    const float*            dy,
    float*                  dx,
    CUDAContext*            ctx) {
    _EdgePad1DGrad<float>
        << < CUDA_BLOCKS(count), CUDA_THREADS,
             0, ctx->cuda_stream() >> >(count,
                 dim, ex_dim, inner_dim, pad_l, dy, dx);
}

/******************** ndarray.one_hot ********************/

template <typename T>
__global__ void _OneHot(
    const int               count,
    const int               depth,
    const int               on_value,
    const float*            x,
    float*                  y) {
    CUDA_1D_KERNEL_LOOP(idx, count) {
        const int val = x[idx];
        y[idx * depth + val] = on_value;
    }
}

template <> void OneHot<float, CUDAContext>(
    const int               count,
    const int               depth,
    const int               on_value,
    const float*            x,
    float*                  y,
    CUDAContext*            ctx) {
    _OneHot<float>
        << < CUDA_BLOCKS(count), CUDA_THREADS,
             0, ctx->cuda_stream() >> >(count,
                 depth, on_value, x, y);
}

/******************** ndarray.reduce ********************/

template <typename T>
__global__ void _Sum(
    const int               count,
    const int               axis_dim,
    const int               inner_dim,
    const T*                x,
    float*                  y) {
    CUDA_1D_KERNEL_LOOP(idx, count) {
        T sum_val = 0.0;
        const int offset = (idx / inner_dim * axis_dim)
            * inner_dim + idx % inner_dim;
        for (int j = 0; j < axis_dim; j++)
            sum_val += x[offset + j * inner_dim];
        y[idx] = sum_val;
   }
}

template<> void Sum<float, CUDAContext>(
    const int               count,
    const int               axis_dim,
    const int               inner_dim,
    const float*            x,
    float*                  y,
    CUDAContext*            ctx) {
    _Sum<float>
        << < CUDA_BLOCKS(count), CUDA_THREADS,
             0, ctx->cuda_stream() >> >(count,
                 axis_dim, inner_dim, x, y);
}

template <typename T>
__global__ void _SumGrad(
    const int               count,
    const int               axis_dim,
    const int               inner_dim,
    const T                 coeff,
    const T*                dy,
    float*                  dx) {
    CUDA_1D_KERNEL_LOOP(idx, count) {
        const int offset = (idx / inner_dim * axis_dim)
            * inner_dim + idx % inner_dim;
        for (int j = 0; j < axis_dim; j++)
            dx[offset + j * inner_dim] = dy[idx] * coeff;
    }
}

template<> void SumGrad<float, CUDAContext>(
    const int               count,
    const int               axis_dim,
    const int               inner_dim,
    const float             coeff,
    const float*            dy,
    float*                  dx,
    CUDAContext*            ctx) {
    _SumGrad<float>
        << < CUDA_BLOCKS(count), CUDA_THREADS,
             0, ctx->cuda_stream() >> >(count,
                 axis_dim, inner_dim, coeff, dy, dx);
}

/******************** ndarray.repeat ********************/

template <typename T>
__global__ void _Repeat(
    const int               count,
    const int               inner_dim,
    const int               repeats,
    const int               dim,
    const T*                x,
    T*                      y) {
    CUDA_1D_KERNEL_LOOP(idx, count) {
        const int d = idx % inner_dim;
        const int b = (idx / inner_dim / repeats) % dim;
        const int n = idx / inner_dim / repeats / dim;
        const int x_idx = (n * dim + b) * inner_dim + d;
        y[idx] = x[x_idx];
    }
}

template <> void Repeat<float, CUDAContext>(
    const int               count,
    const int               outer_dim,
    const int               dim,
    const int               inner_dim,
    const int               repeats,
    const float*            x,
    float*                  y,
    CUDAContext*            ctx) {
    _Repeat<float>
        << < CUDA_BLOCKS(count), CUDA_THREADS,
             0, ctx->cuda_stream() >> >(count,
                 inner_dim, repeats, dim, x, y);
}

template <typename T>
__global__ void _RepeatGrad(
    const int               count,
    const int               inner_dim,
    const int               repeats,
    const int               dim,
    const T*                dy,
    T*                      dx) {
    CUDA_1D_KERNEL_LOOP(idx, count) {
        const int d = idx % inner_dim;
        const int b = (idx / inner_dim) % dim;
        const int n = idx / inner_dim  / dim;
        T gradient = 0;
        for (int t = 0; t < repeats; t++)
            gradient += dy[
                (((n * dim + b) * repeats) + t)
                    * inner_dim + d];
        dx[idx] = gradient;
    }
}

template <> void RepeatGrad<float, CUDAContext>(
    const int               count,
    const int               outer_dim,
    const int               dim,
    const int               inner_dim,
    const int               repeats,
    const float*            dy,
    float*                  dx,
    CUDAContext*            ctx) {
    _RepeatGrad<float>
        << < CUDA_BLOCKS(count), CUDA_THREADS,
             0, ctx->cuda_stream() >> >(
                 count, inner_dim, repeats, dim, dy, dx);
}

/******************** ndarray.slice ********************/

template <typename T>
__global__ void _Slice(
    const int               count,
    const int               outer_dim,
    const int               inner_dim,
    const int               x_slice_dim,
    const int               y_slice_dim,
    const int               slice_offset,
    const T*                x,
    T*                      y) {
    CUDA_1D_KERNEL_LOOP(idx, count) {
        const int tmp = y_slice_dim * inner_dim;
        const int outer_idx = idx / tmp;
        const int slice_idx = idx % tmp;
        const int x_idx = (outer_idx * x_slice_dim + slice_offset)
                                * inner_dim + slice_idx;
        y[idx] = x[x_idx];
    }
}

template <> void Slice<float, CUDAContext>(
    const int               count,
    const int               outer_dim,
    const int               inner_dim,
    const int               x_slice_dim,
    const int               y_slice_dim,
    const int               slice_offset,
    const float*            x,
    float*                  y,
    CUDAContext*            ctx) {
    _Slice<float>
        << < CUDA_BLOCKS(count), CUDA_THREADS,
             0, ctx->cuda_stream() >> >(count,
                 outer_dim, inner_dim,
                    x_slice_dim, y_slice_dim,
                        slice_offset, x, y);
}

template <typename T>
__global__ void _SliceGrad(
    const int               count,
    const int               outer_dim,
    const int               inner_dim,
    const int               x_slice_dim,
    const int               y_slice_dim,
    const int               slice_offset,
    const T*                dy,
    T*                      dx) {
    CUDA_1D_KERNEL_LOOP(idx, count) {
        const int tmp = y_slice_dim * inner_dim;
        const int outer_idx = idx / tmp;
        const int slice_idx = idx % tmp;
        const int x_idx = (outer_idx * x_slice_dim + slice_offset) 
                                * inner_dim + slice_idx;
        dx[x_idx] = dy[idx];
    }
}

template <> void SliceGrad<float, CUDAContext>(
    const int               count,
    const int               outer_dim,
    const int               inner_dim,
    const int               x_slice_dim,
    const int               y_slice_dim,
    const int               slice_offset,
    const float*            dy,
    float*                  dx,
    CUDAContext*            ctx) {
    _SliceGrad<float>
        << < CUDA_BLOCKS(count), CUDA_THREADS,
             0, ctx->cuda_stream() >> >(count,
                 outer_dim, inner_dim,
                     x_slice_dim, y_slice_dim,
                         slice_offset, dy,  dx);
}

/******************** ndarray.tile ********************/

template <typename T>
__global__ void _Tile(
    const int               count,
    const int               ex_inner_dim,
    const int               multiple,
    const T*                x,
    T*                      y) {
    CUDA_1D_KERNEL_LOOP(idx, count) {
        const int d = idx % ex_inner_dim;
        const int n = idx / ex_inner_dim / multiple;
        const int x_idx = n * ex_inner_dim + d;
        y[idx] = x[x_idx];
    }
}

template <> void Tile<float, CUDAContext>(
    const int               count,
    const int               outer_dim,
    const int               ex_inner_dim,
    const int               multiple,
    const float*            x,
    float*                  y,
    CUDAContext*            ctx) {
    _Tile<float>
        << < CUDA_BLOCKS(count), CUDA_THREADS,
             0, ctx->cuda_stream() >> >(count,
                 ex_inner_dim, multiple, x, y);
}

template <typename T>
__global__ void _TileGrad(
    const int               count,
    const int               ex_inner_dim,
    const int               multiple,
    const T*                dy,
    T*                      dx) {
    CUDA_1D_KERNEL_LOOP(idx, count) {
        T gradient = 0;
        const int offset = (idx / ex_inner_dim * multiple)
            * ex_inner_dim + idx % ex_inner_dim;
        for (int t = 0; t < multiple; t++)
            gradient += dy[offset + t * ex_inner_dim];
        dx[idx] = gradient;
    }
}

template <> void TileGrad<float, CUDAContext>(
    const int               count,
    const int               outer_dim,
    const int               ex_inner_dim,
    const int               multiple,
    const float*            dy,
    float*                  dx,
    CUDAContext*            ctx) {
    _TileGrad<float>
        << < CUDA_BLOCKS(count), CUDA_THREADS,
             0, ctx->cuda_stream() >> >(
                 count, ex_inner_dim, multiple, dy, dx);
}

/******************** ndarray.transpose ********************/

template <typename T>
__global__ void _Transpose(
    const int               count,
    const int               ndim,
    const int*              order,
    const int*              old_steps,
    const int*              new_steps,
    const T*                x,
    T*                      y) {
    CUDA_1D_KERNEL_LOOP(idx, count) {
       int x_idx = 0, y_idx = idx;
       for (int j = 0; j < ndim; ++j) {
           int k = order[j];
           x_idx += (y_idx / new_steps[j]) * old_steps[k];
           y_idx %= new_steps[j];
       }
       y[idx] = x[x_idx];
   }
}

template <> void Transpose<float, CUDAContext>(
    const int               count,
    const int               ndim,
    const int*              order,
    const int*              old_steps,
    const int*              new_steps,
    const float*            x,
    float*                  y,
    CUDAContext*            ctx) {
    _Transpose<float>
        << < CUDA_BLOCKS(count), CUDA_THREADS,
             0, ctx->cuda_stream() >> >(count,
                 ndim, order, old_steps, new_steps, x, y);
}

template <typename T>
__global__ void _TransposeGrad(
    const int               count,
    const int               ndim,
    const int*              order,
    const int*              old_steps,
    const int*              new_steps,
    const T*                dy,
    T*                      dx) {
    CUDA_1D_KERNEL_LOOP(idx, count) {
        int x_idx = 0, y_idx = idx;
        for (int j = 0; j < ndim; ++j) {
            int k = order[j];
            x_idx += (y_idx / new_steps[j]) * old_steps[k];
            y_idx %= new_steps[j];
        }
        dx[x_idx] = dy[idx];
    }
}

template <> void TransposeGrad<float, CUDAContext>(
    const int               count,
    const int               ndim,
    const int*              order,
    const int*              old_steps,
    const int*              new_steps,
    const float*            dy,
    float*                  dx,
    CUDAContext*            ctx) {
    _TransposeGrad<float>
        << < CUDA_BLOCKS(count), CUDA_THREADS,
             0, ctx->cuda_stream() >> >(count,
                 ndim, order, old_steps, new_steps, dy, dx);
}

/******************** recurrent.lstm_cell ********************/

template <typename T>
__global__ void _LSTMCellAct(
    const int               count,
    const int               c_offset,
    const int               x_offset,
    T*                      xact) {
    CUDA_1D_KERNEL_LOOP(idx, count) {
        const int offset = idx % x_offset;
        xact[idx] = offset < c_offset ?
            _SigmoidUnit<float>(xact[idx]) : tanh(xact[idx]);
    }
}

template <typename T>
__global__ void _LSTMCellGate(
    const int               count,
    const int               hidden_size,
    const int               o_offset, // 2 * hidden_size
    const int               c_offset, // 3 * hidden_size
    const int               x_offset, // 4 * hidden_size
    const T*                cx,
    const T*                xact,
    T*                      c,
    T*                      h) {
    CUDA_1D_KERNEL_LOOP(idx, count) {
        const int n = idx / hidden_size;
        const int offset = idx % hidden_size;
        const T* x  = xact + n * x_offset;
        const T i = x[offset];
        const T f = x[offset + hidden_size];
        const T o = x[offset + o_offset];
        T c_ = x[offset + c_offset];
        c_ = c[idx] = f * cx[idx] + i * c_;
        h[idx] = o * tanh(c_);
    }
}

template <> void LSTMCell<float, CUDAContext>(
    const int               count,
    const int               N,
    const int               C,
    const float*            cx,
    float*                  xact,
    float*                  c,
    float*                  h,
    CUDAContext*            ctx) {
    const int o_offset = 2 * C,
                  c_offset = 3 * C,
                      x_offset = 4 * C;
    _LSTMCellAct<float>
        << < CUDA_BLOCKS(count * 4), CUDA_THREADS,
             0, ctx->cuda_stream() >> > (count * 4,
                 c_offset, x_offset, xact);
    _LSTMCellGate<float>
        << < CUDA_BLOCKS(count), CUDA_THREADS,
             0, ctx->cuda_stream() >> >(count,
                 C, o_offset, c_offset, x_offset,
                    cx, xact, c, h);
}

template <typename T>
__global__ void _LSTMCellGateGrad(
    const int               count,
    const int               hidden_size,
    const int               o_offset,
    const int               c_offset,
    const int               x_offset,
    const T*                cx,
    const T*                xact,
    const T*                c,
    const T*                dc,
    const T*                dh,
    T*                      dcx,
    T*                      dx) {
    CUDA_1D_KERNEL_LOOP(idx, count) {
        const int n = idx / hidden_size;
        const int offset = idx % hidden_size;
        const T* xact_ = xact + n * x_offset;
        T* dx_ = dx + n * x_offset;
        const T i = xact_[offset];
        const T f = xact_[offset + hidden_size];
        const T o = xact_[offset + o_offset];
        const T g = xact_[offset + c_offset];
        const T tanh_c = tanh(c[idx]);
        const T dcx_sum_term =
            dh[idx] * o * (1 - tanh_c * tanh_c) + dc[idx];
        dcx[idx] = dcx_sum_term * f;
        dx_[offset] = dcx_sum_term * g;
        dx_[offset + hidden_size] = dcx_sum_term * cx[idx];
        dx_[offset + o_offset] = dh[idx] * tanh_c;
        dx_[offset + c_offset] = dcx_sum_term * i;
    }
}

template <typename T>
__global__ void _LSTMCellActGrad(
    const int               count,
    const int               c_offset,
    const int               x_offset,
    const T*                xact,
    T*                      dx) {
    CUDA_1D_KERNEL_LOOP(idx, count) {
        const int offset = idx % x_offset;
        const T val = xact[idx];
        if (offset < c_offset) dx[idx] = dx[idx] * val * (T(1) - val);
        else dx[idx] = dx[idx] * (T(1) - val * val);
    }
}

template <> void LSTMCellGrad<float, CUDAContext>(
    const int               count,
    const int               N,
    const int               C,
    const float*            cx,
    const float*            xact,
    const float*            c,
    const float*            dc,
    const float*            dh,
    float*                  dcx,
    float*                  dx,
    CUDAContext*            ctx) {
    const int o_offset = 2 * C, 
                  c_offset = 3 * C,
                      x_offset = 4 * C;
    _LSTMCellGateGrad<float>
        << < CUDA_BLOCKS(count), CUDA_THREADS,
             0, ctx->cuda_stream() >> >(count,
                 C, o_offset, c_offset, x_offset,
                    cx, xact, c, dc, dh, dcx, dx);
    _LSTMCellActGrad<float>
        << < CUDA_BLOCKS(count * 4), CUDA_THREADS,
             0, ctx->cuda_stream() >> >(count * 4,
                 c_offset, x_offset, xact, dx);
}

/******************** update.adam_update ********************/

template <typename T>
__global__ void _AdamUpdate(
    const int               count,
    const T                 lr,
    const T                 beta1,
    const T                 beta2,
    const T                 eps,
    T*                      g,
    T*                      m,
    T*                      v) {
    CUDA_1D_KERNEL_LOOP(i, count) {
        T gi = g[i];
        T mi = m[i] = m[i] * beta1 + gi * (1 - beta1);
        T vi = v[i] = v[i] * beta2 + gi * gi * (1 - beta2);
        g[i] = lr * mi / (sqrt(vi) + eps);
    }
}

template <> void AdamUpdate<float, CUDAContext>(
    const int               count,
    const float             lr,
    const float             beta1,
    const float             beta2,
    const float             eps,
    float*                  g,
    float*                  m,
    float*                  v,
    CUDAContext*            ctx) {
    _AdamUpdate<float>
        << < CUDA_BLOCKS(count), CUDA_THREADS,
             0, ctx->cuda_stream() >> > (count,
                 lr, beta1, beta2, eps, g, m, v);
}

/******************** update.nesterov_update ********************/

template <typename T>
__global__ void _NesterovUpdate(
    const int               count,
    const T                 lr,
    const T                 momentum,
    T*                      g,
    T*                      h) {
    CUDA_1D_KERNEL_LOOP(i, count) {
        T hi = h[i];
        T hi_new = h[i] = momentum * hi + lr * g[i];
        g[i] = (1 + momentum) * hi_new - momentum * hi;
    }
}

template <> void NesterovUpdate<float, CUDAContext>(
    const int               count,
    const float             lr,
    const float             momentum,
    float*                  g,
    float*                  h,
    CUDAContext*            ctx) {
    _NesterovUpdate<float>
        << < CUDA_BLOCKS(count), CUDA_THREADS,
             0, ctx->cuda_stream() >> > (count,
                 lr, momentum, g, h);
}

/******************** update.rmsprop_update ********************/

template <typename T>
__global__ void _RMSPropUpdate(
    const int               count,
    const T                 lr,
    const T                 decay,
    const T                 eps,
    T*                      g,
    T*                      h) {
    CUDA_1D_KERNEL_LOOP(i, count) {
        T gi = g[i];
        T hi = h[i] = decay * h[i] + (1 - decay) * gi * gi;
        g[i] = lr * g[i] / (sqrt(hi) + eps);
    }
}

template <> void RMSPropUpdate<float, CUDAContext>(
    const int               count,
    const float             lr,
    const float             decay,
    const float             eps,
    float*                  g,
    float*                  h,
    CUDAContext*            ctx) {
    _RMSPropUpdate<float>
        << < CUDA_BLOCKS(count), CUDA_THREADS,
             0, ctx->cuda_stream() >> >(count,
                 lr, decay, eps, g, h);
}

/******************** update.sgd_update ********************/

template <typename T>
__global__ void _SGDUpdate(
    const int               count,
    const T                 lr,
    const T                 momentum,
    T*                      g,
    T*                      h) {
    CUDA_1D_KERNEL_LOOP(i, count) {
        T hi = h[i];
        g[i] = h[i] = momentum * hi + lr * g[i];
    }
}

template <> void SGDUpdate<float, CUDAContext>(
    const int               count,
    const float             lr,
    const float             momentum,
    float*                  g,
    float*                  h,
    CUDAContext*            ctx) {
    _SGDUpdate<float>
        << < CUDA_BLOCKS(count), CUDA_THREADS,
             0, ctx->cuda_stream() >> >(count,
                 lr, momentum, g, h);
}

/******************** vision.bias_add ********************/

template <typename T>
__global__ void _BiasAdd_NCHW(
    const int               count,
    const int               dim,
    const int               inner_dim,
    const T*                bias,
    T*                      y) {
    CUDA_1D_KERNEL_LOOP(idx, count) {
        y[idx] += bias[(idx / inner_dim) % dim];
    }
}

template <typename T>
__global__ void _BiasAdd_NHWC(
    const int               count,
    const int               dim,
    const int               inner_dim,
    const T*                bias,
    T*                      y) {
    CUDA_1D_KERNEL_LOOP(idx, count) {
        y[idx] += bias[idx % dim];
    }
}

template<> void BiasAdd<float, CUDAContext>(
    const int               count,
    const int               outer_dim,
    const int               dim,
    const int               inner_dim,
    const string&           data_format,
    const float*            bias,
    const float*            bias_multiplier,
    float*                  y,
    CUDAContext*            ctx) {
    if (data_format == "NCHW") {
        _BiasAdd_NCHW<float>
            << < CUDA_BLOCKS(count), CUDA_THREADS,
                 0, ctx->cuda_stream() >> >(
                     count, dim, inner_dim, bias, y);
    } else if (data_format == "NHWC") {
        _BiasAdd_NHWC<float>
            << < CUDA_BLOCKS(count), CUDA_THREADS,
                 0, ctx->cuda_stream() >> >(
                     count, dim, inner_dim, bias, y);
    } else LOG(FATAL) << "Unknown data format: " << data_format;
}

/******************** vision.bilinear_resize ********************/

template <typename T>
__global__ void _BilinearResize_NCHW(
    const int               count,
    const int               N,
    const int               C,
    const int               H,
    const int               W,
    const int               out_h,
    const int               out_w,
    const float             scale_h,
    const float             scale_w,
    const T*                x,
    T*                      y) {
    CUDA_1D_KERNEL_LOOP(idx, count) {
        const int w = idx % out_w;
        const int h = (idx / out_w) % out_h;
        const int c = (idx / out_w / out_h) % C;
        const int n = idx / out_w / out_w / C;

        const float h_in = h * scale_h;
        const int top_y_idx = floorf(h_in);
        const int bottom_y_idx = (h_in < H - 1) ? ceilf(h_in) : H - 1;
        const float y_lerp = h_in - top_y_idx;

        const float w_in = w * scale_w;
        const int left_x_idx = floorf(w_in);
        const int right_x_idx = (w_in < W - 1) ? ceilf(w_in) : W - 1;
        const float x_lerp = w_in - left_x_idx;

        const int NCHT = (n * C + c) * H + top_y_idx;
        const int NCHB = (n * C + c) * H + bottom_y_idx;

        const float top_left(x[NCHT * W + left_x_idx]);
        const float top_right(x[NCHT * W + right_x_idx]);
        const float bottom_left(x[NCHB * W + left_x_idx]);
        const float bottom_right(x[NCHB * W + right_x_idx]);

        const float top = top_left + (top_right - top_left) * x_lerp;
        const float bottom = bottom_left + (bottom_right - bottom_left) * x_lerp;
        y[idx] = top + (bottom - top) * y_lerp;
    }
}

template <typename T>
__global__ void _BilinearResize_NHWC(
    const int               count,
    const int               N,
    const int               C,
    const int               H,
    const int               W,
    const int               out_h,
    const int               out_w,
    const float             scale_h,
    const float             scale_w,
    const T*                x,
    T*                      y) {
    CUDA_1D_KERNEL_LOOP(idx, count) {
        const int c = idx % C;
        const int w = (idx / C) % out_w;
        const int h = (idx / C / out_w) % out_h;
        const int n = idx / C / out_w / out_h;

        const float h_in = h * scale_h;
        const int top_y_idx = floorf(h_in);
        const int bottom_y_idx = (h_in < H - 1) ? ceilf(h_in) : H - 1;
        const float y_lerp = h_in - top_y_idx;

        const float w_in = w * scale_w;
        const int left_x_idx = floorf(w_in);
        const int right_x_idx = (w_in < W - 1) ? ceilf(w_in) : W - 1;
        const float x_lerp = w_in - left_x_idx;

        const int NHT = n * H + top_y_idx;
        const int NHB = n * H + bottom_y_idx;

        const float top_left(x[(NHT * W + left_x_idx) * C + c]);
        const float top_right(x[(NHT * W + right_x_idx) * C + c]);
        const float bottom_left(x[(NHB * W + left_x_idx) * C + c]);
        const float bottom_right(x[(NHB * W + right_x_idx) * C + c]);

        const float top = top_left + (top_right - top_left) * x_lerp;
        const float bottom = bottom_left + (bottom_right - bottom_left) * x_lerp;
        y[idx] = top + (bottom - top) * y_lerp;
    }
}

template <> void BilinearResize<float, CUDAContext>(
    const int               count,
    const int               N,
    const int               C,
    const int               H,
    const int               W,
    const int               out_h,
    const int               out_w,
    const string&           data_format,
    const float*            x,
    float*                  y,
    CUDAContext*            ctx) {
    const float scale_h = (float)H / out_h;
    const float scale_w = (float)W / out_w;
     if (data_format == "NCHW") {
         _BilinearResize_NCHW<float>
             << < CUDA_BLOCKS(count), CUDA_THREADS,
                  0, ctx->cuda_stream() >> >(
                      count, N, C, H, W, out_h, out_w,
                          scale_h, scale_w, x, y);
    } else if(data_format == "NHWC") {
         _BilinearResize_NHWC<float>
             << < CUDA_BLOCKS(count), CUDA_THREADS,
                  0, ctx->cuda_stream() >> >(
                      count, N, C, H, W, out_h, out_w,
                          scale_h, scale_w, x, y);
    } else LOG(FATAL) << "Unknown data format: " << data_format;
}

template <typename T>
__global__ void _BilinearResizeGrad_NCHW(
    const int               count,
    const int               N,
    const int               C,
    const int               H,
    const int               W,
    const int               out_h,
    const int               out_w,
    const float             scale_h,
    const float             scale_w,
    const T*                dy,
    T*                      dx) {
    CUDA_1D_KERNEL_LOOP(idx, count) {
        const int w = idx % out_w;
        const int h = (idx / out_w) % out_h;
        const int c = (idx / out_w / out_h) % C;
        const int n = idx / out_w / out_w / C;

        const float h_in = h * scale_h;
        const int top_y_idx = floorf(h_in);
        const int bottom_y_idx = (h_in < H - 1) ? ceilf(h_in) : H - 1;
        const float y_lerp = h_in - top_y_idx;

        const float w_in = w * scale_w;
        const int left_x_idx = floorf(w_in);
        const int right_x_idx = (w_in < W - 1) ? ceilf(w_in) : W - 1;
        const float x_lerp = w_in - left_x_idx;

        const int NCHT = (n * C + c) * H + top_y_idx;
        const int NCHB = (n * C + c) * H + bottom_y_idx;
        const float dtop = (1 - y_lerp) * dy[idx];
        const float dbottom = y_lerp * dy[idx];

        atomicAdd(&dx[NCHT * W + left_x_idx], static_cast<T>((1 - x_lerp) * dtop));
        atomicAdd(&dx[NCHT * W + right_x_idx], static_cast<T>(x_lerp * dtop));
        atomicAdd(&dx[NCHB * W + left_x_idx], static_cast<T>((1 - x_lerp) * dbottom));
        atomicAdd(&dx[NCHB * W + right_x_idx], static_cast<T>(x_lerp * dbottom));
    }
}

template <typename T>
__global__ void _BilinearResizeGrad_NHWC(
    const int               count,
    const int               N,
    const int               C,
    const int               H,
    const int               W,
    const int               out_h,
    const int               out_w,
    const float             scale_h,
    const float             scale_w,
    const T*                dy,
    T*                      dx) {
    CUDA_1D_KERNEL_LOOP(idx, count) {
        const int c = idx % C;
        const int w = (idx / C) % out_w;
        const int h = (idx / C / out_w) % out_h;
        const int n = idx / C / out_w / out_h;

        const float h_in = h * scale_h;
        const int top_y_idx = floorf(h_in);
        const int bottom_y_idx = (h_in < H - 1) ? ceilf(h_in) : H - 1;
        const float y_lerp = h_in - top_y_idx;

        const float w_in = w * scale_w;
        const int left_x_idx = floorf(w_in);
        const int right_x_idx = (w_in < W - 1) ? ceilf(w_in) : W - 1;
        const float x_lerp = w_in - left_x_idx;

        const int NHT = n * H + top_y_idx;
        const int NHB = n * H + bottom_y_idx;
        const float dtop = (1 - y_lerp) * dy[idx];
        const float dbottom = y_lerp * dy[idx];

        atomicAdd(&dx[(NHT * W + left_x_idx) * C + c], static_cast<T>((1 - x_lerp) * dtop));
        atomicAdd(&dx[(NHT * W + right_x_idx) * C + c], static_cast<T>(x_lerp * dtop));
        atomicAdd(&dx[(NHB * W + left_x_idx) * C + c], static_cast<T>((1 - x_lerp) * dbottom));
        atomicAdd(&dx[(NHB * W + right_x_idx) * C + c], static_cast<T>(x_lerp * dbottom));
    }
}

template <> void BilinearResizeGrad<float, CUDAContext>(
    const int               count,
    const int               N,
    const int               C,
    const int               H,
    const int               W,
    const int               out_h,
    const int               out_w,
    const string&           data_format,
    const float*            dy,
    float*                  dx,
    CUDAContext*            ctx) {
    const float scale_h = (float)H / out_h;
    const float scale_w = (float)W / out_w;
     if (data_format == "NCHW") {
         _BilinearResizeGrad_NCHW<float>
             << < CUDA_BLOCKS(count), CUDA_THREADS,
                  0, ctx->cuda_stream() >> >(
                      count, N, C, H, W, out_h, out_w,
                          scale_h, scale_w, dy, dx);
    } else if(data_format == "NHWC") {
         _BilinearResizeGrad_NHWC<float>
             << < CUDA_BLOCKS(count), CUDA_THREADS,
                  0, ctx->cuda_stream() >> >(
                      count, N, C, H, W, out_h, out_w,
                          scale_h, scale_w, dy, dx);
    } else LOG(FATAL) << "Unknown data format: " << data_format;
}

/******************** vision.conv ********************/

template<typename T>
__global__ void _Im2Col2d_NCHW(
    const int               count,
    const int               H,
    const int               W,
    const int               col_h,
    const int               col_w,
    const int               kernel_h,
    const int               kernel_w,
    const int               stride_h,
    const int               stride_w,
    const int               pad_h,
    const int               pad_w,
    const int               dilation_h,
    const int               dilation_w,
    const T*                im,
    T*                      col) {
    CUDA_1D_KERNEL_LOOP(idx, count) {
        const int w = idx % col_w;
        const int h_idx = idx / col_w;
        const int h = h_idx % col_h;
        const int im_c = h_idx / col_h;
        const int c = im_c * kernel_h * kernel_w;

        const int im_h_off = h * stride_h - pad_h;
        const int im_w_off = w * stride_w - pad_w;

        T* col_ptr = col;
        col_ptr += ((c * col_h + h) * col_w + w);

        const T* im_ptr = im;
        im_ptr += ((im_c * H + im_h_off) * W + im_w_off);

        for (int kh = 0; kh < kernel_h; kh++) {
            for (int kw = 0; kw < kernel_w; kw++) {
                const int im_h = kh * dilation_h + im_h_off;
                const int im_w = kw * dilation_w + im_w_off;
                *col_ptr = (im_h >= 0 && im_w >= 0 && im_h < H && im_w < W) ? 
                    im_ptr[kh * dilation_h * W + kw * dilation_w] : 0;
                col_ptr += (col_h * col_w);
            }
        }
    }
}

template<typename T>
__global__ void _Im2Col2d_NHWC(
    const int               count,
    const int               C,
    const int               H,
    const int               W,
    const int               col_h,
    const int               col_w,
    const int               kernel_h,
    const int               kernel_w,
    const int               stride_h,
    const int               stride_w,
    const int               pad_h,
    const int               pad_w,
    const int               dilation_h,
    const int               dilation_w,
    const T*                im,
    T*                      col) {
    CUDA_1D_KERNEL_LOOP(idx, count) {
        const int c = idx % C;
        const int w = (idx / C) % col_w;
        const int h = idx / C / col_w;
      
        const int im_h_off = h * stride_h - pad_h;
        const int im_w_off = w * stride_w - pad_w;
        const int base_col_idx = (h * col_w) + w;

        for (int kh = 0; kh < kernel_h; kh++) {
            for (int kw = 0; kw < kernel_w; kw++) {
                const int im_h = kh * dilation_h + im_h_off;
                const int im_w = kw * dilation_w + im_w_off;
                const int col_idx = (
                    ((base_col_idx * kernel_h + kh) * kernel_w + kw) * C + c
                );
                col[col_idx] = (im_h >= 0 && im_w >= 0 &&
                    im_h < H && im_w < W) ? im[(im_h * W + im_w) * C + c] : 0;
            }
        }
    }
}

template <> void Im2Col2d<float, CUDAContext>(
    const int               C,
    const int               H,
    const int               W,
    const int               col_h,
    const int               col_w,
    const int               kernel_h,
    const int               kernel_w,
    const int               stride_h,
    const int               stride_w,
    const int               pad_h,
    const int               pad_w,
    const int               dilation_h,
    const int               dilation_w,
    const string&           data_format,
    const float*            im,
    float*                  col,
    CUDAContext*            ctx) {
    if (data_format == "NCHW") {
         const int count = (C * col_h * col_w);
         _Im2Col2d_NCHW<float> 
             << < CUDA_BLOCKS(count), CUDA_THREADS,
                  0, ctx->cuda_stream() >> >(count,
                      H, W, col_h, col_w, kernel_h, kernel_w,
                          stride_h, stride_w, pad_h, pad_w,
                              dilation_h, dilation_w, im, col);
    } else if (data_format == "NHWC") {
         const int count = (col_h * col_w * C);
         _Im2Col2d_NHWC<float> 
             << < CUDA_BLOCKS(count), CUDA_THREADS,
                  0, ctx->cuda_stream() >> >(count, 
                      C, H, W, col_h, col_w, kernel_h, kernel_w,
                          stride_h, stride_w, pad_h, pad_w,
                              dilation_h, dilation_w, im, col);
    } else LOG(FATAL) << "Unknown data format: " << data_format;
}

template<typename T>
__global__ void _Col2Im2d_NCHW(
    const int               count,
    const int               H,
    const int               W,
    const int               col_h,
    const int               col_w,
    const int               kernel_h,
    const int               kernel_w,
    const int               stride_h,
    const int               stride_w,
    const int               pad_h,
    const int               pad_w,
    const int               dilation_h,
    const int               dilation_w,
    const T*                col,
    T*                      im) {
    CUDA_1D_KERNEL_LOOP(idx, count) {
        T val = 0;
        const int im_w = idx % W + pad_w;
        const int im_h = (idx / W) % H + pad_h;
        const int im_c = idx / W / H;
        const int ex_kernel_h = (kernel_h - 1) * dilation_h + 1;
        const int ex_kernel_w = (kernel_w - 1) * dilation_w + 1;

        //  redundant pixels will be ignored when conv
        //  note to clip them by min(x,col_w)
        const int w_start = (im_w < ex_kernel_w) ?
            0 : (im_w - ex_kernel_w) / stride_w + 1;
        const int w_end = min(im_w / stride_w + 1, col_w);
        const int h_start = (im_h < ex_kernel_h) ? 
            0 : (im_h - ex_kernel_h) / stride_h + 1;
        const int h_end = min(im_h / stride_h + 1, col_h);

        for (int h = h_start; h < h_end; ++h) {
            for (int w = w_start; w < w_end; ++w) {
                int kh_off = (im_h - h * stride_h);
                int kw_off = (im_w - w * stride_w);
                //  only the serval im pixels used in dilated-conv
                //  ignore the corresponding col pixels
                if (kh_off % dilation_h == 0 && kw_off % dilation_w == 0) {
                    kh_off /= dilation_h;
                    kw_off /= dilation_w;
                    const int col_idx = ((
                        (im_c * kernel_h + kh_off) * kernel_w + kw_off) * col_h + h
                    ) * col_w + w;
                    val += col[col_idx];
                }
            }
        }
        im[idx] = val;
    }
}

template<typename T>
__global__ void _Col2Im2d_NHWC(
    const int               count,
    const int               C,
    const int               H,
    const int               W,
    const int               col_h,
    const int               col_w,
    const int               kernel_h,
    const int               kernel_w,
    const int               stride_h,
    const int               stride_w,
    const int               pad_h,
    const int               pad_w,
    const int               dilation_h,
    const int               dilation_w,
    const T*                col,
    T*                      im) {
    CUDA_1D_KERNEL_LOOP(idx, count) {
        T val = 0;
        const int im_c = idx % C;
        const int im_w = (idx / C) % W + pad_w;
        const int im_h = (idx / C / W) + pad_h;
        const int ex_kernel_h = (kernel_h - 1) * dilation_h + 1;
        const int ex_kernel_w = (kernel_w - 1) * dilation_w + 1;

        //  redundant pixels will be ignored when conv
        //  note to clip them by min(x,col_w)
        const int w_start = (im_w < ex_kernel_w) ?
            0 : (im_w - ex_kernel_w) / stride_w + 1;
        const int w_end = min(im_w / stride_w + 1, col_w);
        const int h_start = (im_h < ex_kernel_h) ?
            0 : (im_h - ex_kernel_h) / stride_h + 1;
        const int h_end = min(im_h / stride_h + 1, col_h);

        for (int h = h_start; h < h_end; ++h) {
            for (int w = w_start; w < w_end; ++w) {
                int kh_off = (im_h - h * stride_h);
                int kw_off = (im_w - w * stride_w);
                //  only the serval im pixels used in dilated-conv
                //  ignore the corresponding col pixels
                if (kh_off % dilation_h == 0 && kw_off % dilation_w == 0) {
                    kh_off /= dilation_h;
                    kw_off /= dilation_w;
                    const int col_idx = (
                        ((h * col_w + w) * kernel_h + kh_off) * kernel_w + kw_off
                    ) * C + im_c;
                    val += col[col_idx];
                }
            }
        }
        im[idx] = val;
    }
}

template <> void Col2Im2d<float, CUDAContext>(
    const int               C,
    const int               H,
    const int               W,
    const int               col_h,
    const int               col_w,
    const int               kernel_h,
    const int               kernel_w,
    const int               stride_h,
    const int               stride_w,
    const int               pad_h,
    const int               pad_w,
    const int               dilation_h,
    const int               dilation_w,
    const string&           data_format,
    const float*            col,
    float*                  im,
    CUDAContext*            ctx) {
    if (data_format == "NCHW") {
         const int count = (C * H * W);
         _Col2Im2d_NCHW<float>
             << < CUDA_BLOCKS(count), CUDA_THREADS,
                  0, ctx->cuda_stream() >> >(count,
                      H, W, col_h, col_w, kernel_h, kernel_w,
                          stride_h, stride_w, pad_h, pad_w,
                              dilation_h, dilation_w, col, im);
    } else if (data_format == "NHWC") {
         const int count = (H * W * C);
         _Col2Im2d_NHWC<float> 
             << < CUDA_BLOCKS(count), CUDA_THREADS,
                  0, ctx->cuda_stream() >> >(count,
                      C, H, W, col_h, col_w, kernel_h, kernel_w,
                          stride_h, stride_w, pad_h, pad_w,
                              dilation_h, dilation_w, col, im);
    } else LOG(FATAL) << "Unknown data format: " << data_format;
}

/******************** vision.nn_resize ********************/

template <typename T>
__global__ void _NNResize_NCHW(
    const int               count,
    const int               N,
    const int               C,
    const int               H,
    const int               W,
    const int               out_h,
    const int               out_w,
    const float             scale_h,
    const float             scale_w,
    const T*                x,
    T*                      y) {
    CUDA_1D_KERNEL_LOOP(idx, count) {
        const int w = idx % out_w;
        const int h = (idx / out_w) % out_h;
        const int c = (idx / out_w / out_h) % C;
        const int n = idx / out_w / out_h / C;
        const int h_in = min(int(floorf(h * scale_h)), H - 1);
        const int w_in = min(int(floorf(w * scale_w)), W - 1);
        y[idx] = x[((n * C + c) * H + h_in) * W + w_in];
    }
}

template <typename T>
__global__ void _NNResize_NHWC(
    const int               count,
    const int               N,
    const int               C,
    const int               H,
    const int               W,
    const int               out_h,
    const int               out_w,
    const float             scale_h,
    const float             scale_w,
    const T*                x,
    T*                      y) {
    CUDA_1D_KERNEL_LOOP(idx, count) {
        const int c = idx % C;
        const int w = (idx / C) % out_w;
        const int h = (idx / C / out_w) % out_h;
        const int n = idx / C / out_w / out_h;
        const int h_in = min(int(floorf(h * scale_h)), H - 1);
        const int w_in = min(int(floorf(w * scale_w)), W - 1);
        y[idx] = x[((n * H + h_in) * W + w_in) * C + c];
    }
}

template <> void NNResize<float, CUDAContext>(
    const int               count,
    const int               N,
    const int               C,
    const int               H,
    const int               W,
    const int               out_h,
    const int               out_w,
    const string&           data_format,
    const float*            x,
    float*                  y,
    CUDAContext*            ctx) {
    const float scale_h = (float)H / out_h;
    const float scale_w = (float)W / out_w;
    if (data_format == "NCHW") {
        _NNResize_NCHW<float>
            << < CUDA_BLOCKS(count), CUDA_THREADS,
                 0, ctx->cuda_stream() >> >(
                     count, N, C, H, W, out_h, out_w,
                         scale_h, scale_w, x, y);
    } else if(data_format == "NHWC") {
        _NNResize_NHWC<float>
            << < CUDA_BLOCKS(count), CUDA_THREADS,
                 0, ctx->cuda_stream() >> >(
                     count, N, C, H, W, out_h, out_w,
                         scale_h, scale_w, x, y);
    } else LOG(FATAL) << "Unknown data format: " << data_format;
}

template <typename T>
 __global__ void _NNResizeGrad_NCHW(
     const int              count,
     const int              N,
     const int              C,
     const int              H,
     const int              W,
     const int              out_h,
     const int              out_w,
     const float            scale_h,
     const float            scale_w,
     const T*               dy,
     T*                     dx) {
    CUDA_1D_KERNEL_LOOP(idx, count) {
        const int w = idx % out_w;
        const int h = (idx / out_w) % out_h;
        const int c = (idx / out_w / out_h) % C;
        const int n = idx / out_w / out_h / C;
        const int h_in = min(int(floorf(h * scale_h)), H - 1);
        const int w_in = min(int(floorf(w * scale_w)), W - 1);
        atomicAdd(&dx[((n * C + c) * H + h_in) * W + w_in], dy[idx]);
    }
}

template <typename T>
__global__ void _NNResizeGrad_NHWC(
    const int               count,
    const int               N,
    const int               C,
    const int               H,
    const int               W,
    const int               out_h,
    const int               out_w,
    const float             scale_h,
    const float             scale_w,
    const T*                dy,
    T*                      dx) {
    CUDA_1D_KERNEL_LOOP(idx, count) {
        const int c = idx % C;
        const int w = (idx / C) % out_w;
        const int h = (idx / C / out_w) % out_h;
        const int n = idx / C / out_w / out_h;
        const int h_in = min(int(floorf(h * scale_h)), H - 1);
        const int w_in = min(int(floorf(w * scale_w)), W - 1);
        atomicAdd(&dx[((n * H + h_in) * W + w_in) * C + c], dy[idx]);
    }
}

template <> void NNResizeGrad<float, CUDAContext>(
    const int               count,
    const int               N,
    const int               C,
    const int               H,
    const int               W,
    const int               out_h,
    const int               out_w,
    const string&           data_format,
    const float*            dy,
    float*                  dx,
    CUDAContext*            ctx) {
    const float scale_h = (float)H / out_h;
    const float scale_w = (float)W / out_w;
    if (data_format == "NCHW") {
        _NNResizeGrad_NCHW<float>
            << < CUDA_BLOCKS(count), CUDA_THREADS,
                 0, ctx->cuda_stream() >> >(
                     count, N, C, H, W, out_h, out_w,
                         scale_h, scale_w, dy, dx);
    } else if(data_format == "NHWC") {
        _NNResizeGrad_NHWC<float> 
            << < CUDA_BLOCKS(count), CUDA_THREADS,
                 0, ctx->cuda_stream() >> >(
                     count, N, C, H, W, out_h, out_w,
                         scale_h, scale_w, dy, dx);
    } else LOG(FATAL) << "Unknown data format: " << data_format;
}

/******************** vision.pooling ********************/

template<typename T>
__global__ void _MAXPooling2d_NCHW(
    const int               count,
    const int               N,
    const int               C,
    const int               H,
    const int               W,
    const int               pool_h,
    const int               pool_w,
    const int               kernel_h,
    const int               kernel_w,
    const int               stride_h,
    const int               stride_w,
    const int               pad_h,
    const int               pad_w,
    const T*                x,
    int*                    mask,
    T*                      y) {
    CUDA_1D_KERNEL_LOOP(idx, count) {
        const int pw = idx % pool_w;
        const int ph = (idx / pool_w) % pool_h;
        const int pc = (idx / pool_w / pool_h) % C;
        const int pn = idx / pool_w / pool_h / C;

        int start_h = ph * stride_h - pad_h;
        int start_w = pw * stride_w - pad_w;
        const int end_h = min(start_h + kernel_h, H);
        const int end_w = min(start_w + kernel_w, W);

        start_h = max(start_h, 0);
        start_w = max(start_w, 0);

        T max_val = -FLT_MAX;
        int max_idx = -1;
        const T* x_ptr = x + (pn * C + pc) * H * W;

        for (int h = start_h; h < end_h; ++h) {
            for (int w = start_w; w < end_w; ++w) {
                if (x_ptr[h * W + w] > max_val) {
                    max_idx = h * W + w;
                    max_val = x_ptr[max_idx];
                }
            }
        }
        y[idx] = max_val;
        mask[idx] = max_idx;
    }
}

template<typename T>
__global__ void _MAXPooling2d_NHWC(
    const int               count,
    const int               N,
    const int               C,
    const int               H,
    const int               W,
    const int               pool_h,
    const int               pool_w,
    const int               kernel_h,
    const int               kernel_w,
    const int               stride_h,
    const int               stride_w,
    const int               pad_h,
    const int               pad_w,
    const T*                x,
    int*                    mask,
    T*                      y) {
    CUDA_1D_KERNEL_LOOP(idx, count) {
        const int pc = idx % C;
        const int pw = (idx / C) % pool_w;
        const int ph = (idx / C / pool_w) % pool_h;
        const int pn = idx / C / pool_w / pool_h;

        int start_h = ph * stride_h - pad_h;
        int start_w = pw * stride_w - pad_w;
        const int end_h = min(start_h + kernel_h, H);
        const int end_w = min(start_w + kernel_w, W);

        start_h = max(start_h, 0);
        start_w = max(start_w, 0);

        T max_val = -FLT_MAX;
        int max_idx = -1;
        for (int h = start_h; h < end_h; ++h) {
            for (int w = start_w; w < end_w; ++w) {
                const int x_idx = ((pn * H + h) * W + w) * C + pc;
                if (x[x_idx] > max_val) {
                    max_idx = x_idx;
                    max_val = x[max_idx];
                }
            }
        }
        y[idx] = max_val;
        mask[idx] = max_idx;
    }
}

template<> void MAXPooling2d<float, CUDAContext>(
    const int               count,
    const int               N,
    const int               C,
    const int               H,
    const int               W,
    const int               pool_h,
    const int               pool_w,
    const int               kernel_h,
    const int               kernel_w,
    const int               stride_h,
    const int               stride_w,
    const int               pad_h,
    const int               pad_w,
    const string&           data_format,
    const float*            x,
    int*                    mask,
    float*                  y,
    CUDAContext*            ctx) {
    if (data_format == "NCHW") {
        _MAXPooling2d_NCHW<float>
            << < CUDA_BLOCKS(count), CUDA_THREADS,
                 0, ctx->cuda_stream() >> >(count,
                     N, C, H, W, pool_h, pool_w, kernel_h, kernel_w,
                         stride_h, stride_w, pad_h, pad_w, x, mask, y);
    } else if (data_format == "NHWC") {                        
        _MAXPooling2d_NHWC<float>
            << < CUDA_BLOCKS(count), CUDA_THREADS,
                 0, ctx->cuda_stream() >> >(count,
                     N, C, H, W, pool_h, pool_w, kernel_h, kernel_w,
                         stride_h, stride_w, pad_h, pad_w, x, mask, y);
    } else LOG(FATAL) << "Unknown data format: " << data_format;
}

template<typename T>
__global__ void _AVGPooling2d_NCHW(
    const int               count,
    const int               N,
    const int               C,
    const int               H,
    const int               W,
    const int               pool_h,
    const int               pool_w,
    const int               kernel_h,
    const int               kernel_w,
    const int               stride_h,
    const int               stride_w,
    const int               pad_h,
    const int               pad_w,
    const T*                x,
    T*                      y) {
    CUDA_1D_KERNEL_LOOP(idx, count) {
        const int pw = idx % pool_w;
        const int ph = (idx / pool_w) % pool_h;
        const int pc = (idx / pool_w / pool_h) % C;
        const int pn = idx / pool_w / pool_h / C;

        int start_h = ph * stride_h - pad_h;
        int start_w = pw * stride_w - pad_w;
        int end_h = min(start_h + kernel_h, H + pad_h);
        int end_w = min(start_w + kernel_w, W + pad_w);

        start_h = max(start_h, 0);
        start_w = max(start_w, 0);
        end_h = min(end_h, H);
        end_w = min(end_w, W);

        const T* x_ptr = x + (pn * C + pc) * H * W;
        const int pool_area = (end_h - start_h) * (end_w - start_w);
        T avg_val = 0;

        for (int h = start_h; h < end_h; ++h) {
            for (int w = start_w; w < end_w; ++w) {
                avg_val += x_ptr[h * W + w];
            }
        }
        y[idx] = avg_val / pool_area;
    }
}

template<typename T>
__global__ void _AVGPooling2d_NHWC(
    const int               count,
    const int               N,
    const int               C,
    const int               H,
    const int               W,
    const int               pool_h,
    const int               pool_w,
    const int               kernel_h,
    const int               kernel_w,
    const int               stride_h,
    const int               stride_w,
    const int               pad_h,
    const int               pad_w,
    const T*                x,
    T*                      y) {
    CUDA_1D_KERNEL_LOOP(idx, count) {
        const int pc = idx % C;
        const int pw = (idx / C) % pool_w;
        const int ph = (idx / C / pool_w) % pool_h;
        const int pn = idx / C / pool_w / pool_h;

        int start_h = ph * stride_h - pad_h;
        int start_w = pw * stride_w - pad_w;
        int end_h = min(start_h + kernel_h, H + pad_h);
        int end_w = min(start_w + kernel_w, W + pad_w);

        start_h = max(start_h, 0);
        start_w = max(start_w, 0);
        end_h = min(end_h, H);
        end_w = min(end_w, W);

        const int pool_area = (end_h - start_h) * (end_w - start_w);
        T avg_val = 0;

        for (int h = start_h; h < end_h; ++h) 
            for (int w = start_w; w < end_w; ++w)
                avg_val += x[((pn * H + h) * W + w) * C + pc];

        y[idx] = avg_val / pool_area;
    }
}

template<> void AVGPooling2d<float, CUDAContext>(
    const int               count,
    const int               N,
    const int               C,
    const int               H,
    const int               W,
    const int               pool_h,
    const int               pool_w,
    const int               kernel_h,
    const int               kernel_w,
    const int               stride_h,
    const int               stride_w,
    const int               pad_h,
    const int               pad_w,
    const string&           data_format,
    const float*            x,
    float*                  y,
    CUDAContext*            ctx) {
    if (data_format == "NCHW") {
        _AVGPooling2d_NCHW<float>
            << < CUDA_BLOCKS(count), CUDA_THREADS,
                 0, ctx->cuda_stream() >> >(count,
                     N, C, H, W, pool_h, pool_w, kernel_h, kernel_w,
                         stride_h, stride_w, pad_h, pad_w, x, y);
    } else if (data_format == "NHWC") {
        _AVGPooling2d_NHWC<float>
            << < CUDA_BLOCKS(count), CUDA_THREADS,
                 0, ctx->cuda_stream() >> >(count,
                     N, C, H, W, pool_h, pool_w, kernel_h, kernel_w,
                         stride_h, stride_w, pad_h, pad_w, x, y);
    } else LOG(FATAL) << "Unknown data format: " << data_format;
}

template<typename T>
__global__ void _MAXPooling2dGrad_NCHW(
    const int               count,
    const int               N,
    const int               C,
    const int               H,
    const int               W,
    const int               pool_h,
    const int               pool_w,
    const int               kernel_h,
    const int               kernel_w,
    const int               stride_h,
    const int               stride_w,
    const int               pad_h,
    const int               pad_w,
    const T*                dy,
    const int*              mask,
    T*                      dx) {
    CUDA_1D_KERNEL_LOOP(idx, count) {
        const int w = idx % W;
        const int h = (idx / W) % H;
        const int c = (idx / W / H) % C;
        const int n = idx / W / H / C;

        //  allow overlapping
        const int start_ph = (h + pad_h < kernel_h) ?
            0 : (h + pad_h - kernel_h) / stride_h + 1;
        const int start_pw = (w + pad_w < kernel_w) ? 
            0 : (w + pad_w - kernel_w) / stride_w + 1;
        //  allow clip
        const int end_ph = min((h + pad_h) / stride_h + 1, pool_h);
        const int end_pw = min((w + pad_w) / stride_w + 1, pool_w);

        T grad = 0;
        const int offset = (n * C + c) * pool_h * pool_w;
        const T* dy_ptr = dy + offset;
        const int* mask_ptr = mask + offset;

        for (int ph = start_ph; ph < end_ph; ++ph) {
            for (int pw = start_pw; pw < end_pw; ++pw) {
                if (mask_ptr[ph * pool_w + pw] == (h * W + w)) {
                    grad += dy_ptr[ph * pool_w + pw];
                }
            }
        }
        dx[idx] = grad;
    }
}

template<typename T>
__global__ void _MAXPooling2dGrad_NHWC(
    const int               count,
    const int               N,
    const int               C,
    const int               H,
    const int               W,
    const int               pool_h,
    const int               pool_w,
    const int               kernel_h,
    const int               kernel_w,
    const int               stride_h,
    const int               stride_w,
    const int               pad_h,
    const int               pad_w,
    const T*                dy,
    const int*              mask,
    T*                      dx) {
    CUDA_1D_KERNEL_LOOP(idx, count) {
        const int c = idx % C;
        const int w = (idx / C) % W;
        const int h = (idx / C / W) % H;
        const int n = idx / C / W / H;

        //  allow overlapping
        const int start_ph = (h + pad_h < kernel_h) ?
            0 : (h + pad_h - kernel_h) / stride_h + 1;
        const int start_pw = (w + pad_w < kernel_w) ?
            0 : (w + pad_w - kernel_w) / stride_w + 1;
        //  allow clip
        const int end_ph = min((h + pad_h) / stride_h + 1, pool_h);
        const int end_pw = min((w + pad_w) / stride_w + 1, pool_w);

        T grad = 0;
        for (int ph = start_ph; ph < end_ph; ++ph) {
            for (int pw = start_pw; pw < end_pw; ++pw) {
                const int x_idx = ((n * H + h) * W + w) * C + c;
                const int y_idx = ((n * pool_h + ph) * pool_w + pw) * C + c;
                if (mask[y_idx] == x_idx) grad += dy[y_idx];
            }
        }
        dx[idx] = grad;
    }
}

template<> void MAXPooling2dGrad<float, CUDAContext>(
    const int               count,
    const int               N,
    const int               C,
    const int               H,
    const int               W,
    const int               pool_h,
    const int               pool_w,
    const int               kernel_h,
    const int               kernel_w,
    const int               stride_h,
    const int               stride_w,
    const int               pad_h,
    const int               pad_w,
    const string&           data_format,
    const float*            dy,
    const int*              mask,
    float*                  dx,
    CUDAContext*            ctx) {
    if (data_format == "NCHW") {
        _MAXPooling2dGrad_NCHW<float>
            << < CUDA_BLOCKS(count), CUDA_THREADS,
                 0, ctx->cuda_stream() >> >(count,
                     N, C, H, W, pool_h, pool_w, kernel_h, kernel_w,
                         stride_h, stride_w, pad_h, pad_w, dy,mask, dx);
    } else if (data_format == "NHWC") {
        _MAXPooling2dGrad_NHWC<float>
            << < CUDA_BLOCKS(count), CUDA_THREADS,
                 0, ctx->cuda_stream() >> >(count,
                     N, C, H, W, pool_h, pool_w, kernel_h, kernel_w,
                         stride_h, stride_w, pad_h, pad_w, dy, mask, dx);
    } else LOG(FATAL) << "Unknown data format: " << data_format;
}

template<typename T>
__global__ void _AVGPooling2dGrad_NCHW(
    const int               count,
    const int               N,
    const int               C,
    const int               H,
    const int               W,
    const int               pool_h,
    const int               pool_w,
    const int               kernel_h,
    const int               kernel_w,
    const int               stride_h,
    const int               stride_w,
    const int               pad_h,
    const int               pad_w,
    const T*                dy,
    T*                      dx) {
    CUDA_1D_KERNEL_LOOP(idx, count) {
        const int w = idx % W;
        const int h = (idx / W) % H;
        const int c = (idx / W / H) % C;
        const int n = idx / W / H / C;

        const int start_ph = (h + pad_h < kernel_h) ?
            0 : (h + pad_h - kernel_h) / stride_h + 1;
        const int start_pw = (w + pad_w < kernel_w) ?
            0 : (w + pad_w - kernel_w) / stride_w + 1;
        const int end_ph = min(h / stride_h + 1, pool_h);
        const int end_pw = min(w / stride_w + 1, pool_w);

        T grad = 0;
        const T* dy_ptr = dy + (n * C + c) * pool_h * pool_w;
        for (int ph = start_ph; ph < end_ph; ++ph) {
            for (int pw = start_pw; pw < end_pw; ++pw) {
                int start_h = ph * stride_h - pad_h;
                int start_w = pw * stride_w - pad_w;
                int end_h = min(start_h + kernel_h, H + pad_h);
                int end_w = min(start_w + kernel_w, W + pad_w);
                int pool_area = (end_h - start_h) * (end_w - start_w);
                grad += (dy_ptr[ph * pool_w + pw] / pool_area);
            }
        }
        dx[idx] = grad;
    }
}

template<typename T>
__global__ void _AVGPooling2dGrad_NHWC(
    const int               count,
    const int               N,
    const int               C,
    const int               H,
    const int               W,
    const int               pool_h,
    const int               pool_w,
    const int               kernel_h,
    const int               kernel_w,
    const int               stride_h,
    const int               stride_w,
    const int               pad_h,
    const int               pad_w,
    const T*                dy,
    T*                      dx) {
    CUDA_1D_KERNEL_LOOP(idx, count) {
        const int c = idx % C;
        const int w = (idx / C) % W;
        const int h = (idx / C / W) % H;
        const int n = idx / C / W / H;

        const int start_ph = (h + pad_h < kernel_h) ?
            0 : (h + pad_h - kernel_h) / stride_h + 1;
        const int start_pw = (w + pad_w < kernel_w) ?
            0 : (w + pad_w - kernel_w) / stride_w + 1;
        const int end_ph = min(h / stride_h + 1, pool_h);
        const int end_pw = min(w / stride_w + 1, pool_w);

        T grad = 0;
        for (int ph = start_ph; ph < end_ph; ++ph) {
            for (int pw = start_pw; pw < end_pw; ++pw) {
                int start_h = ph * stride_h - pad_h;
                int start_w = pw * stride_w - pad_w;
                int end_h = min(start_h + kernel_h, H + pad_h);
                int end_w = min(start_w + kernel_w, W + pad_w);
                int pool_area = (end_h - start_h) * (end_w - start_w);
                const int y_idx = ((n * pool_h + ph) * pool_w + pw) * C + c;
                grad += (dy[y_idx] / pool_area);
            }
        }
        dx[idx] = grad;
    }
}

template<> void AVGPooling2dGrad<float, CUDAContext>(
    const int               count,
    const int               N,
    const int               C,
    const int               H,
    const int               W,
    const int               pool_h,
    const int               pool_w,
    const int               kernel_h,
    const int               kernel_w,
    const int               stride_h,
    const int               stride_w,
    const int               pad_h,
    const int               pad_w,
    const string&           data_format,
    const float*            dy,
    float*                  dx,
    CUDAContext*            ctx) {
   if (data_format == "NCHW") {
        _AVGPooling2dGrad_NCHW<float>
            << < CUDA_BLOCKS(count), CUDA_THREADS,
                 0, ctx->cuda_stream() >> >(count,
                     N, C, H, W, pool_h, pool_w, kernel_h, kernel_w,
                         stride_h, stride_w, pad_h, pad_w, dy, dx);
    } else if (data_format == "NHWC") {
        _AVGPooling2dGrad_NHWC<float>
            << < CUDA_BLOCKS(count), CUDA_THREADS,
                 0, ctx->cuda_stream() >> >(count,
                     N, C, H, W, pool_h, pool_w, kernel_h, kernel_w,
                         stride_h, stride_w, pad_h, pad_w, dy, dx);
    } else LOG(FATAL) << "Unknown data format: " << data_format;
}

/******************** vision.roi_pooling ********************/

template <typename T>
__global__ void _ROIPooling(
    const int               count,
    const T                 spatial_scale,
    const int               channels,
    const int               height,
    const int               width,
    const int               pool_h,
    const int               pool_w,
    const T*                x,
    const T*                rois,
    int*                    mask,
    T*                      y) {
    CUDA_1D_KERNEL_LOOP(idx, count) {
        int pw = idx % pool_w;
        int ph = (idx / pool_w) % pool_h;
        int c = (idx / pool_w / pool_h) % channels;
        int n = idx / pool_w / pool_h / channels;

        const T* offset_rois = rois + n * 5;
        int roi_batch_ind = offset_rois[0];

        if (roi_batch_ind < 0) {
            y[idx] = 0;
            mask[idx] = -1;
            continue;
        }

        int roi_start_w = round(offset_rois[1] * spatial_scale);
        int roi_start_h = round(offset_rois[2] * spatial_scale);
        int roi_end_w = round(offset_rois[3] * spatial_scale);
        int roi_end_h = round(offset_rois[4] * spatial_scale);

        int roi_width = max(roi_end_w - roi_start_w + 1, 1);
        int roi_height = max(roi_end_h - roi_start_h + 1, 1);
        const T bin_size_h = (T)roi_height / (T)pool_h;
        const T bin_size_w = (T)roi_width / (T)pool_w;

        int hstart = floor(bin_size_h * ph);
        int wstart = floor(bin_size_w * pw);
        int hend = ceil(bin_size_h * (ph + 1));
        int wend = ceil(bin_size_w * (pw + 1));

        hstart = min(max(hstart + roi_start_h, 0), height);
        hend = min(max(hend + roi_start_h, 0), height);
        wstart = min(max(wstart + roi_start_w, 0), width);
        wend = min(max(wend + roi_start_w, 0), width);

        bool is_empty = (hend <= hstart) || (wend <= wstart);
        float max_val = is_empty ? 0 : -FLT_MAX;
        int max_idx = -1;
        x += ((roi_batch_ind * channels + c) * height * width);
        for (int h = hstart; h < hend; ++h) {
            for (int w = wstart; w < wend; ++w) {
                const int x_idx = h * width + w;
                if (x[x_idx] > max_val) {
                    max_val = x[x_idx];
                    max_idx = x_idx;
                }
            }
        }
        y[idx] = max_val;
        mask[idx] = max_idx;
    }
}

template<> void ROIPooling<float, CUDAContext>(
    const int               count,
    const int               N,
    const int               C,
    const int               H,
    const int               W,
    const int               pool_h,
    const int               pool_w,
    const int               num_rois,
    const float             spatial_scale,
    const float*            x,
    const float*            rois,
    int*                    mask,
    float*                  y,
    CUDAContext*            ctx) {
    _ROIPooling<float>
        << < CUDA_BLOCKS(count), CUDA_THREADS,
             0, ctx->cuda_stream() >> >(count,
                 spatial_scale, C, H, W,
                     pool_h, pool_w, x, rois, mask, y);
}

template <typename T>
__global__ void _ROIPoolingGrad(
    const int               count,
    const int               num_rois,
    const T                 spatial_scale,
    const int               channels,
    const int               height,
    const int               width,
    const int               pool_h,
    const int               pool_w,
    const T*                dy,
    const T*                rois,
    const int*              mask,
    T*                      dx) {
    CUDA_1D_KERNEL_LOOP(idx, count) {
        int w = idx % width;
        int h = (idx / width) % height;
        int c = (idx / width / height) % channels;
        int n = idx / width / height / channels;

        T gradient = 0;

        for (int roi_n = 0; roi_n < num_rois; ++roi_n) {
            const T* offset_rois = rois + roi_n * 5;
            int roi_batch_ind = offset_rois[0];

            if (n != roi_batch_ind) continue;

            int roi_start_w = round(offset_rois[1] * spatial_scale);
            int roi_start_h = round(offset_rois[2] * spatial_scale);
            int roi_end_w = round(offset_rois[3] * spatial_scale);
            int roi_end_h = round(offset_rois[4] * spatial_scale);

            const bool in_roi = (w >= roi_start_w &&
                                 w <= roi_end_w &&
                                 h >= roi_start_h &&
                                 h <= roi_end_h);

            if (!in_roi) continue;

            int y_offset = (roi_n * channels + c) * pool_h * pool_w;
            const T* offset_dy = dy + y_offset;
            const int* offset_mask = mask + y_offset;

            int roi_width = max(roi_end_w - roi_start_w + 1, 1);
            int roi_height = max(roi_end_h - roi_start_h + 1, 1);

            const T bin_size_h = (T)roi_height / (T)pool_h;
            const T bin_size_w = (T)roi_width / (T)pool_w;

            int phstart = floor(static_cast<T>(h - roi_start_h) / bin_size_h);
            int phend = ceil(static_cast<T>(h - roi_start_h + 1) / bin_size_h);
            int pwstart = floor(static_cast<T>(w - roi_start_w) / bin_size_w);
            int pwend = ceil(static_cast<T>(w - roi_start_w + 1) / bin_size_w);

            phstart = min(max(phstart, 0), pool_h);
            phend = min(max(phend, 0), pool_h);
            pwstart = min(max(pwstart, 0), pool_w);
            pwend = min(max(pwend, 0), pool_w);

            for (int ph = phstart; ph < phend; ++ph) {
                for (int pw = pwstart; pw < pwend; ++pw) {
                    int pool_idx = ph * pool_w + pw;
                    if (offset_mask[pool_idx] == (h * width + w)) {
                        gradient += offset_dy[pool_idx];
                    }
                }
            }
        }
        dx[idx] = gradient;
    }
}

template<> void ROIPoolingGrad<float, CUDAContext>(
    const int               count,
    const int               N,
    const int               C,
    const int               H,
    const int               W,
    const int               pool_h,
    const int               pool_w,
    const int               num_rois,
    const float             spatial_scale,
    const float*            dy,
    const float*            rois,
    const int*              mask,
    float*                  dx,
    CUDAContext*            ctx) {
    _ROIPoolingGrad<float>
        << < CUDA_BLOCKS(count), CUDA_THREADS,
             0, ctx->cuda_stream() >> >(count,
                 num_rois, spatial_scale, C, H, W,
                     pool_h, pool_w, dy, rois, mask, dx);
}

/******************** vision.roi_align ********************/

template <typename T>
__device__ T _ROIAlignInterpolate(
    const T*                Xdata,
    const int               height,
    const int               width,
    T                       y,
    T                       x) {
    if (y < -1.0 || y > height || x < -1.0 || x > width) return 0;
    if (y <= 0) y = 0;
    if (x <= 0) x = 0;

    int y_low = (int)y;
    int x_low = (int)x;
    int y_high;
    int x_high;

    if (y_low >= height - 1) {
        y_high = y_low = height - 1;
        y = (T)y_low;
    } else {
        y_high = y_low + 1;
    }

    if (x_low >= width - 1) {
        x_high = x_low = width - 1;
        x = (T)x_low;
    } else {
        x_high = x_low + 1;
    }

    T ly = y - y_low;
    T lx = x - x_low;
    T hy = 1. - ly, hx = 1. - lx;
    T v1 = Xdata[y_low * width + x_low];
    T v2 = Xdata[y_low * width + x_high];
    T v3 = Xdata[y_high * width + x_low];
    T v4 = Xdata[y_high * width + x_high];
    T w1 = hy * hx, w2 = hy * lx, w3 = ly * hx, w4 = ly * lx;
    T val = (w1 * v1 + w2 * v2 + w3 * v3 + w4 * v4);
    return val;
}

template <typename T>
__global__ void _ROIAlign(
    const int               count,
    const float             spatial_scale,
    const int               channels,
    const int               height,
    const int               width,
    const int               pool_h,
    const int               pool_w,
    const int               sampling_ratio,
    const T*                Xdata,
    const T*                rois,
    T*                      Ydata) {
    CUDA_1D_KERNEL_LOOP(idx, count) {
        int pw = idx % pool_w;
        int ph = (idx / pool_w) % pool_h;
        int c = (idx / pool_w / pool_h) % channels;
        int n = idx / pool_w / pool_h / channels;

        const T* offset_rois = rois + n * 5;
        int roi_batch_ind = offset_rois[0];

        if (roi_batch_ind < 0) {
            Ydata[idx] = 0;
            continue;
        }

        T roi_start_w = offset_rois[1] * spatial_scale;
        T roi_start_h = offset_rois[2] * spatial_scale;
        T roi_end_w = offset_rois[3] * spatial_scale;
        T roi_end_h = offset_rois[4] * spatial_scale;

        T roi_width = max(roi_end_w - roi_start_w, (T)1.);
        T roi_height = max(roi_end_h - roi_start_h, (T)1.);
        T bin_size_h = static_cast<T>(roi_height) / static_cast<T>(pool_h);
        T bin_size_w = static_cast<T>(roi_width) / static_cast<T>(pool_w);

        const T* offset_Xdata = Xdata +(roi_batch_ind * channels + c) * height * width;

        int roi_bin_grid_h = (sampling_ratio > 0) ?
            sampling_ratio : ceil(roi_height / pool_h);
        int roi_bin_grid_w = (sampling_ratio > 0) ? 
            sampling_ratio : ceil(roi_width / pool_w);

        const T num_bin_grids = roi_bin_grid_h * roi_bin_grid_w;

        T output_val = 0.;
        for (int iy = 0; iy < roi_bin_grid_h; iy++) {
            const T y = roi_start_h + ph * bin_size_h +
                static_cast<T>(iy + .5f) * bin_size_h / static_cast<T>(roi_bin_grid_h);
            for (int ix = 0; ix < roi_bin_grid_w; ix++) {
                const T x = roi_start_w + pw * bin_size_w + 
                    static_cast<T>(ix + .5f) * bin_size_w / static_cast<T>(roi_bin_grid_w);
                T val = _ROIAlignInterpolate(offset_Xdata, height, width, y, x);
                output_val += val;
            }
        }
        output_val /= num_bin_grids;
        Ydata[idx] = output_val;
    }
}

template<> void ROIAlign<float, CUDAContext>(
    const int               count,
    const int               N,
    const int               C,
    const int               H,
    const int               W,
    const int               pool_h,
    const int               pool_w,
    const int               num_rois,
    const float             spatial_scale,
    const int               sampling_ratio,
    const float*            x,
    const float*            rois,
    float*                  y,
    CUDAContext*            ctx) {
    _ROIAlign<float>
        << < CUDA_BLOCKS(count), CUDA_THREADS,
             0, ctx->cuda_stream() >> >(count,
                 spatial_scale, C, H, W, pool_h, pool_w,
                     sampling_ratio, x, rois, y);
}

template <typename T>
__device__ void _ROIAlignInterpolateGrad(
    const int               height,
    const int               width,
    T                       y,
    T                       x,
    T&                      w1,
    T&                      w2,
    T&                      w3,
    T&                      w4,
    int&                    x_low,
    int&                    x_high,
    int&                    y_low,
    int&                    y_high) {
    if (y < -1.0 || y > height ||
            x < -1.0 || x > width) {
        w1 = w2 = w3 = w4 = 0.;
        x_low = x_high = y_low = y_high = -1;
        return;
    }

    if (y <= 0) y = 0;
    if (x <= 0) x = 0;

    y_low = (int)y;
    x_low = (int)x;

    if (y_low >= height - 1) {
        y_high = y_low = height - 1;
        y = (T)y_low;
    } else {
        y_high = y_low + 1;
    }

    if (x_low >= width - 1) {
        x_high = x_low = width - 1;
        x = (T)x_low;
    } else {
        x_high = x_low + 1;
    }

    T ly = y - y_low;
    T lx = x - x_low;
    T hy = 1. - ly, hx = 1. - lx;
    w1 = hy * hx, w2 = hy * lx, w3 = ly * hx, w4 = ly * lx;
    return;
}

template <typename T>
__global__ void _ROIAlignGrad(
    const int               count,
    const int               num_rois,
    const T                 spatial_scale,
    const int               channels,
    const int               height,
    const int               width,
    const int               pool_h,
    const int               pool_w,
    const int               sampling_ratio,
    const T*                dYdata,
    const T*                rois,
    T*                      dXdata) {
    CUDA_1D_KERNEL_LOOP(idx, count) {
        int pw = idx % pool_w;
        int ph = (idx / pool_w) % pool_h;
        int c = (idx / pool_w / pool_h) % channels;
        int n = idx / pool_w / pool_h / channels;

        const T* offset_rois = rois + n * 5;
        int roi_batch_ind = offset_rois[0];

        if (roi_batch_ind < 0) continue;

        T roi_start_w = offset_rois[1] * spatial_scale;
        T roi_start_h = offset_rois[2] * spatial_scale;
        T roi_end_w = offset_rois[3] * spatial_scale;
        T roi_end_h = offset_rois[4] * spatial_scale;

        T roi_width = max(roi_end_w - roi_start_w, (T)1.);
        T roi_height = max(roi_end_h - roi_start_h, (T)1.);
        T bin_size_h = static_cast<T>(roi_height) / static_cast<T>(pool_h);
        T bin_size_w = static_cast<T>(roi_width) / static_cast<T>(pool_w);

        T* offset_dXdata = dXdata + 
            (roi_batch_ind * channels + c) * height * width;

        int y_offset = (n * channels + c) * pool_h * pool_w;
        const T* offset_dYdata = dYdata + y_offset;
        const T dYdata_this_bin = offset_dYdata[ph * pool_w + pw];

        int roi_bin_grid_h = (sampling_ratio > 0) ?
            sampling_ratio : ceil(roi_height / pool_h);
        int roi_bin_grid_w = (sampling_ratio > 0) ?
            sampling_ratio : ceil(roi_width / pool_w);

        const T num_bin_grids = roi_bin_grid_h * roi_bin_grid_w;

        for (int iy = 0; iy < roi_bin_grid_h; iy++) {
            const T y = roi_start_h + ph * bin_size_h +
                static_cast<T>(iy + .5f) * bin_size_h /
                    static_cast<T>(roi_bin_grid_h);
            for (int ix = 0; ix < roi_bin_grid_w; ix++) {
                const T x = roi_start_w + pw * bin_size_w +
                    static_cast<T>(ix + .5f) * bin_size_w /
                        static_cast<T>(roi_bin_grid_w);

                T w1, w2, w3, w4;
                int x_low, x_high, y_low, y_high;

                _ROIAlignInterpolateGrad(
                    height, width, y, x, w1, w2, w3, w4,
                        x_low, x_high, y_low, y_high);

                T g1 = dYdata_this_bin * w1 / num_bin_grids;
                T g2 = dYdata_this_bin * w2 / num_bin_grids;
                T g3 = dYdata_this_bin * w3 / num_bin_grids;
                T g4 = dYdata_this_bin * w4 / num_bin_grids;

                if (x_low >= 0 && x_high >= 0 
                        && y_low >= 0 && y_high >= 0) {
                    atomicAdd(
                        offset_dXdata + y_low * width + x_low,
                        static_cast<T>(g1));
                    atomicAdd(
                        offset_dXdata + y_low * width + x_high,
                        static_cast<T>(g2));
                    atomicAdd(
                        offset_dXdata + y_high * width + x_low,
                        static_cast<T>(g3));
                    atomicAdd(
                        offset_dXdata + y_high * width + x_high,
                        static_cast<T>(g4));
                }
            }
        }
    }
}

template<> void ROIAlignGrad<float, CUDAContext>(
    const int               count,
    const int               N,
    const int               C,
    const int               H,
    const int               W,
    const int               pool_h,
    const int               pool_w,
    const int               num_rois,
    const float             spatial_scale,
    const int               sampling_ratio,
    const float*            dy,
    const float*            rois,
    float*                  dx,
    CUDAContext*            ctx) {
    _ROIAlignGrad<float>
        << < CUDA_BLOCKS(count), CUDA_THREADS,
             0, ctx->cuda_stream() >> >(count,
                 num_rois, spatial_scale, C, H, W,
                     pool_h, pool_w, sampling_ratio, dy, rois, dx);
}

}    // namespace kernel

}    // namespace dragon

#endif // WITH_CUDA